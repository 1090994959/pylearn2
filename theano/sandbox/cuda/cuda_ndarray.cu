#include "hip/hip_runtime.h"
#include <Python.h>
#include <structmember.h>

#include <numpy/arrayobject.h>
#include <iostream>

#include "cuda_ndarray.cuh"
#ifndef DONT_UNROLL
#define UNROLL_LOOP
#endif

#ifndef SHARED_SIZE 
#define SHARED_SIZE (16*1024)
#endif
/////////////////////////
// Static helper methods
/////////////////////////

template <typename T>
static T ceil_intdiv(T a, T b)
{
    return (a/b) + ((a % b) ? 1: 0);
}

static void
CudaNdarray_null_init(CudaNdarray*self)
{
    self->base = NULL;
    self->nd = -1;
    self->host_structure = NULL;
    self->data_allocated = 0;
    self->dev_structure_fresh = 1;
    self->dev_structure = NULL;
    self->devdata = NULL;
}

static int
CudaNdarray_uninit(CudaNdarray*self)
{
    int rval = 0;
    if (self->data_allocated) {
        assert(self->devdata);
        cublasFree(self->devdata);
        if (HIPBLAS_STATUS_SUCCESS != cublasGetError())
        {
            std::cerr << "!!!! error freeing device memory\n";
            rval = -1;
        }
        self->devdata = NULL;
        self->data_allocated = 0;
    }
    if (self->dev_structure)
    {
        cublasFree(self->dev_structure);
        if (HIPBLAS_STATUS_SUCCESS != cublasGetError())
        {
            std::cerr << "!!!! error freeing device memory\n";
            rval = -1;
        }
        self->dev_structure = NULL;
    }
    if (self->host_structure)
    {
        free(self->host_structure);
        self->host_structure = NULL;
    }
    self->nd = -1;
    Py_XDECREF(self->base);
    self->base = NULL;
    return rval;
}


//make the rightmost coords change fastest 
//TODO: why does a downward for-loop not work????
//TODO: use the log2_dims and driver code to remove / and %
//TODO: skip the last division (when d == 0) 
#define decl_k_elemwise_unary_rowmajor(name, F) \
__global__ void name (unsigned int numEls,  \
        unsigned int nd, \
        const int * dim,  \
        const float * a_data, const int * a_str, \
        float * z_data, const int * z_str) \
{ \
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; \
    const unsigned int numThreads = blockDim.x * gridDim.x; \
 \
    for (unsigned int i = idx; i < numEls; i += numThreads) \
    { \
        unsigned int ii = i; \
        const float * a_i = a_data; \
        float * z_i = z_data; \
        for (unsigned int _d = 0; _d < nd; ++_d) \
        { \
            unsigned int d = nd - _d-1;  \
            unsigned int i_d = ii % dim[d]; /* i_d is our position in the d'th dimension   */ \
            ii = ii / dim[d]; \
            a_i += i_d * a_str[d]; /* increment our a and z pointers by i_d elements */ \
            z_i += i_d * z_str[d]; \
        } \
        z_i[0] = F(a_i[0]); \
    } \
} 

template<typename T> __device__ T unary_copy(T a) { return a; }
decl_k_elemwise_unary_rowmajor(k_elemwise_unary_rowmajor_copy, unary_copy<float>)

template<typename T> __device__ T unary_exp(T a) { return exp(a); }
decl_k_elemwise_unary_rowmajor(k_elemwise_unary_rowmajor_exp, unary_exp<float>)

/////////////////////////////
// Satisfying reqs to be Type
/////////////////////////////

static void
CudaNdarray_dealloc(CudaNdarray* self)
{
    //std::cerr << "CudaNdarray dealloc " << self << " " << self->devdata << '\n';
    CudaNdarray_uninit(self);
    self->ob_type->tp_free((PyObject*)self);
}

static PyObject *
CudaNdarray_new(PyTypeObject *type, PyObject *args, PyObject *kwds)
{
    CudaNdarray *self;

    self = (CudaNdarray *)type->tp_alloc(type, 0);
    if (self != NULL)
    {
        CudaNdarray_null_init(self);
    }
    return (PyObject *)self;
}
static int
CudaNdarray_init(CudaNdarray *self, PyObject *args, PyObject *kwds)
{
    PyObject *arr=NULL;

    if (! PyArg_ParseTuple(args, "O", &arr))
        return -1; 
    if (! PyArray_Check(arr))
    {
        PyErr_SetString(PyExc_TypeError, "PyArray arg required");
        return -1;
    }
    int rval = CudaNdarray_CopyFromArray(self, (PyArrayObject*)arr);
    return rval;
}
static PyMemberDef CudaNdarray_members[] = 
{
    /*
    {"first", T_OBJECT_EX, offsetof(CudaNdarray, first), 0,
     "first name"},
    {"last", T_OBJECT_EX, offsetof(CudaNdarray, last), 0,
     "last name"},
    {"number", T_INT, offsetof(CudaNdarray, number), 0,
     "noddy number"},
     */
    {NULL}  /* Sentinel */
};

PyObject * CudaNdarray_CreateArrayObj(CudaNdarray * self)
{
    int verbose = 0;
    if ((self->nd < 0) || (self->devdata == 0))
    {
        PyErr_SetString(PyExc_ValueError, "can't copy from un-initialized CudaNdarray");
        return NULL;
    }
    CudaNdarray * contiguous_self = NULL;
    if (CudaNdarray_is_c_contiguous(self))
    {
        contiguous_self = self;
        Py_INCREF(contiguous_self);
        if (verbose) std::cerr << "CreateArrayObj already contiguous" << contiguous_self << '\n';
    }
    else
    {
        contiguous_self = (CudaNdarray*)CudaNdarray_Copy(self);
        if (verbose) std::cerr << "CreateArrayObj created contiguous" << contiguous_self << '\n';
    }
    if (!contiguous_self)
    {
        return NULL;
    }

    npy_intp * npydims = (npy_intp*)malloc(self->nd * sizeof(npy_intp));
    assert (npydims);
    for (int i = 0; i < self->nd; ++i) npydims[i] = (npy_intp)(CudaNdarray_HOST_DIMS(self)[i]);
    PyObject * rval = PyArray_SimpleNew(self->nd, npydims, REAL_TYPENUM);
    free(npydims);
    if (!rval)
    {
        Py_DECREF(contiguous_self);
        return NULL;
    }

    assert (PyArray_ITEMSIZE(rval) == sizeof(real));

    hipblasGetVector(PyArray_SIZE(rval), sizeof(real),
            contiguous_self->devdata, 1, 
            PyArray_DATA(rval), 1);
    CNDA_THREAD_SYNC;

    if (HIPBLAS_STATUS_SUCCESS != cublasGetError())
    {
        PyErr_SetString(PyExc_RuntimeError, "error copying data to host");
        Py_DECREF(rval);
        rval = NULL;
    }

    Py_DECREF(contiguous_self);
    return rval;
}
PyObject * CudaNdarray_Copy(CudaNdarray * self)
{
    PyObject * rval = CudaNdarray_new_null();
    if ((!rval) or (-1 == self->nd))
    {
        return rval;
    }
    if (CudaNdarray_alloc_contiguous((CudaNdarray*)rval, self->nd, CudaNdarray_HOST_DIMS(self)))
    {
        Py_DECREF(rval);
        return NULL;
    }
    if (CudaNdarray_CopyFromCudaNdarray((CudaNdarray*)rval, self))
    {
        Py_DECREF(rval);
        return NULL;
    }
    return rval;
}
PyObject * CudaNdarray_DeepCopy(CudaNdarray * self, PyObject * memo)
{
    assert(PyDict_Check(memo));
    PyObject * selfkey = PyInt_FromLong((long)self);
    assert(selfkey);
    if (PyDict_Contains(memo, selfkey))
    {
        PyObject * rval = PyDict_GetItem(memo, selfkey);
        Py_DECREF(selfkey);
        Py_XINCREF(rval);
        return rval;
    }
    else
    {
        PyObject * rval = CudaNdarray_Copy(self);
        //std::cerr << "DeepCopy created " << rval << " devdata " << ((CudaNdarray*)rval)->devdata << "\n";
        if (NULL == rval)
        {
            Py_DECREF(selfkey);
            return NULL;
        }
        if (PyDict_SetItem(memo, selfkey, rval))
        {
            Py_DECREF(rval);
            Py_DECREF(selfkey);
            return NULL;
        }
        Py_DECREF(selfkey);
        return rval;
    }
}
PyObject * CudaNdarray_ReduceSum(CudaNdarray * self, PyObject * py_reduce_mask)
{
    if (!PySequence_Check(py_reduce_mask))
    {
        PyErr_SetString(PyExc_TypeError, "reduce_mask must be sequence of ints");
        return NULL;
    }
    int len = PySequence_Length(py_reduce_mask);
    if (len != self->nd)
    {
        PyErr_SetString(PyExc_TypeError, "length of reduce_mask must match self->nd");
        return NULL;
    }
    CudaNdarray * self_sum = (CudaNdarray*)CudaNdarray_new_null();
    if (!self_sum)
    {
        return NULL;
    }
    int * dimshuffle_pattern = (int*)malloc(len * 2 * sizeof(int));
    int * sum_dims = dimshuffle_pattern + len;
    int n_remaining_dims = 0;
    if (!dimshuffle_pattern)
    {
        Py_DECREF(self_sum);
        PyErr_SetString(PyExc_MemoryError, "failed to alloc internal storage");
        return NULL;
    }
    for (int i = 0; i < len; ++i)
    {
        PyObject *o_i = PySequence_GetItem(py_reduce_mask, i);
        int o_i_int = PyInt_AsLong(o_i);
        Py_XDECREF(o_i);
        if (PyErr_Occurred())
        {
            Py_DECREF(self_sum);
            free(dimshuffle_pattern);
            return NULL;
        }
        if (o_i_int) // this is a dimension over which we are reducing
        {
            sum_dims[i] = 1;
        }
        else
        {
            sum_dims[i] = CudaNdarray_HOST_DIMS(self)[i];
            dimshuffle_pattern[n_remaining_dims++] = i;
        }
    }
    if (0   || CudaNdarray_alloc_contiguous(self_sum, len, sum_dims)
            || CudaNdarray_reduce_sum(self_sum, self)
            || CudaNdarray_dimshuffle(self_sum, n_remaining_dims, dimshuffle_pattern))
    {
        Py_DECREF(self_sum);
        free(dimshuffle_pattern);
        return NULL;
    }
    free(dimshuffle_pattern);
    return (PyObject*)self_sum;
}

__global__ void k_copy_reshape_rowmajor(unsigned int numEls, 
        unsigned int a_nd, const float * a_data, const int * a_dim, const int * a_str,
        unsigned int z_nd, float * z_data, const int * z_dim, const int * z_str)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < numEls; i += numThreads)
    {
        const float * a_i = a_data;
        unsigned int a_ii = i;
        for (unsigned int _d = 0; _d < a_nd; ++_d) //make the rightmost coords change fastest
        {
            unsigned int d = a_nd - _d-1; 
            unsigned int a_i_d = a_ii % a_dim[d];
            a_ii = a_ii / a_dim[d];
            a_i += a_i_d * a_str[d];
        }
        unsigned int z_ii = i;
        float * z_i = z_data;
        for (unsigned int _d = 0; _d < z_nd; ++_d) //make the rightmost coords change fastest
        {
            unsigned int d = z_nd - _d-1; 
            //i tried to make the for loop count down, but it didn't work!?
            unsigned int z_i_d = z_ii % z_dim[d];
            z_i += z_i_d * z_str[d];
            z_ii = z_ii / z_dim[d];
        }
        z_i[0] = a_i[0]; //copy one lousy float!
    }
}
PyObject * CudaNdarray_Reshape(CudaNdarray * self, PyObject * shape)
{
    // check shape tuple
    if (!PyTuple_Check(shape))
    {
        PyErr_SetString(PyExc_TypeError, "shape must be tuple of integers");
        return NULL;
    }
    // copy shape to integer array
    unsigned int rval_nd = PyTuple_Size(shape);
    unsigned int * rval_dims = (unsigned int*)malloc(rval_nd * sizeof(int));
    unsigned int rval_size = 1;
    for (int i = 0; i < rval_nd; ++i)
    {
        rval_dims[i] = PyInt_AsLong(PyTuple_GetItem(shape, i)); //GetItem returns borrowed reference
        if (PyErr_Occurred()) //error in AsLong
        {
            free(rval_dims);
            return NULL;
        }
	if(rval_dims[i]<=0){
	  PyErr_Format(PyExc_ValueError, "Reshape has invalid dimension %i (must be >0)",rval_dims[i]);
	  free(rval_dims);
	  return NULL;
	}
        rval_size = rval_size * rval_dims[i];
    }
    // calculate new size, assert same as old size
    if (rval_size != CudaNdarray_SIZE(self))
    {
        PyErr_SetString(PyExc_ValueError, "size must remain unchanged");
        free(rval_dims);
        return NULL;
    }

    if(CudaNdarray_is_c_contiguous(self))
    {
        //return a view, not a copy
        CudaNdarray * rval = (CudaNdarray * )CudaNdarray_New(rval_nd);
	
	if (!rval || 0 != rval->data_allocated
	    ||CudaNdarray_set_device_data(rval, CudaNdarray_DEV_DATA(self), self))
	  {
	    Py_XDECREF(rval);
	    free(rval_dims);
	    return NULL;
	  }
	//set dim and stride
	int size = 1;
	for (int i = rval_nd-1; i >= 0; --i)
	  {
	    CudaNdarray_set_stride(rval, i, (rval_dims[i] == 1) ? 0 : size);
	    CudaNdarray_set_dim(rval, i, rval_dims[i]);
	    size = size * rval_dims[i];
	  }
	free(rval_dims);
	return (PyObject*)rval;
    }

    // allocate new space (TODO: test to see if we can re-use old one)
    CudaNdarray * rval = (CudaNdarray * )CudaNdarray_new_null();
    if (!rval || CudaNdarray_alloc_contiguous(rval, rval_nd, rval_dims))
    {
        Py_XDECREF(rval);
        free(rval_dims);
        return NULL;
    }

    // call worker routine
    unsigned int threads_per_block = std::min(rval_size, (unsigned int)NUM_VECTOR_OP_THREADS_PER_BLOCK);
    unsigned int n_blocks = std::min(ceil_intdiv(rval_size,threads_per_block), (unsigned int)NUM_VECTOR_OP_BLOCKS);
    k_copy_reshape_rowmajor<<<n_blocks,threads_per_block>>>(
            rval_size, 
            self->nd, 
            CudaNdarray_DEV_DATA(self), CudaNdarray_DEV_DIMS(self), CudaNdarray_DEV_STRIDES(self),
            rval->nd,
            CudaNdarray_DEV_DATA(rval), CudaNdarray_DEV_DIMS(rval), CudaNdarray_DEV_STRIDES(rval));

    CNDA_THREAD_SYNC;
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        Py_DECREF(rval);
        PyErr_Format(PyExc_RuntimeError, "Cuda error: %s: %s.\n", "k_copy_reshape_rowmajor", hipGetErrorString(err));
        free(rval_dims);
        return NULL;
    }                         
    free(rval_dims);
    return (PyObject*)rval;
}
PyObject * CudaNdarray_View(CudaNdarray * self)
{
    CudaNdarray * rval = (CudaNdarray*)CudaNdarray_New(self->nd);
    if (!rval || CudaNdarray_set_device_data(rval, CudaNdarray_DEV_DATA(self), self))
    {
        Py_XDECREF(rval);
        rval = NULL;
    }
    else
    {
        for (int i = 0; i < self->nd; ++i)
        {
            CudaNdarray_set_dim(rval, i, CudaNdarray_HOST_DIMS(self)[i]);
            CudaNdarray_set_stride(rval, i, CudaNdarray_HOST_STRIDES(self)[i]);
        }
    }
    return (PyObject*)rval;
}
PyObject * CudaNdarray_SetStride(CudaNdarray * self, PyObject *args)
{
    int pos, stride;
    if (! PyArg_ParseTuple(args, "ii", &pos, &stride))
        return NULL; 
    if ((pos < 0) || (pos >= self->nd))
    {
        PyErr_Format(PyExc_ValueError, "position argument out of legal range [0, %i)", self->nd);
        return NULL;
    }
    CudaNdarray_set_stride(self, pos, stride);
    if (cnda_copy_structure_to_device(self))
    {
        return NULL;
    }
    Py_INCREF(Py_None);
    return Py_None;
}
PyObject * CudaNdarray_SetShapeI(CudaNdarray * self, PyObject *args)
{
    int pos, dim;
    if (! PyArg_ParseTuple(args, "ii", &pos, &dim))
        return NULL; 
    if ((pos < 0) || (pos >= self->nd))
    {
        PyErr_Format(PyExc_ValueError, "position argument out of legal range [0, %i)", self->nd);
        return NULL;
    }
    CudaNdarray_set_dim(self, pos, dim);
    if (cnda_copy_structure_to_device(self))
    {
        return NULL;
    }
    Py_INCREF(Py_None);
    return Py_None;
}

static PyObject *
CudaNdarray_exp(CudaNdarray* self)
{
    CudaNdarray * rval = (CudaNdarray *)CudaNdarray_new_null();
    if ((NULL == rval) || CudaNdarray_alloc_contiguous(rval, self->nd, CudaNdarray_HOST_DIMS(self)))
    {
        Py_XDECREF(rval);
        return NULL;
    }
    unsigned int size = 1;
    for (int i = 0; i < self->nd; i++)
    {
        size *= (unsigned int) CudaNdarray_HOST_DIMS(self)[i];
    }
    unsigned int threads_per_block = std::min(size, (unsigned int)NUM_VECTOR_OP_THREADS_PER_BLOCK);
    unsigned int n_blocks = std::min(ceil_intdiv(size,threads_per_block), (unsigned int)NUM_VECTOR_OP_BLOCKS);
    k_elemwise_unary_rowmajor_exp<<<n_blocks,threads_per_block>>>(size, self->nd, CudaNdarray_DEV_DIMS(self), 
            CudaNdarray_DEV_DATA(self), CudaNdarray_DEV_STRIDES(self),
            CudaNdarray_DEV_DATA(rval), CudaNdarray_DEV_STRIDES(rval));

    //TODO: don't do this right away, do it when we need the result
    CNDA_THREAD_SYNC;
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        Py_DECREF(rval);
        PyErr_Format(PyExc_RuntimeError, "Cuda error: %s: %s.\n", "kExp", hipGetErrorString(err));
        return NULL;
    }                         

    return (PyObject*)rval;
}
static PyMethodDef CudaNdarray_methods[] = 
{
    {"__array__", 
        (PyCFunction)CudaNdarray_CreateArrayObj, METH_NOARGS,
        "Copy from the device to a numpy ndarray"},
    {"__copy__", 
        (PyCFunction)CudaNdarray_Copy, METH_NOARGS,
        "Create a copy of this object"},
    {"__deepcopy__", 
        (PyCFunction)CudaNdarray_DeepCopy, METH_O,
        "Create a copy of this object"},
    {"reduce_sum",
        (PyCFunction)CudaNdarray_ReduceSum, METH_O,
        "Reduce over the given dimensions by summation"},
    {"exp",
        (PyCFunction)CudaNdarray_exp, METH_NOARGS,
        "Return the exponential of all elements"},
    {"reshape",
        (PyCFunction)CudaNdarray_Reshape, METH_O, 
        "Return a reshaped view (or copy) of this ndarray\n\
            The required argument is a tuple of integers specifying the shape of the new ndarray."},
    {"view",
        (PyCFunction)CudaNdarray_View, METH_NOARGS,
        "Return an alias of this ndarray"},
    {"_set_stride",
        (PyCFunction)CudaNdarray_SetStride, METH_VARARGS,
        "For integer arguments (i, s), set the 'i'th stride to 's'"},
    {"_set_shape_i",
        (PyCFunction)CudaNdarray_SetShapeI, METH_VARARGS,
        "For integer arguments (i, s), set the 'i'th shape to 's'"},
    {NULL, NULL, NULL, NULL}  /* Sentinel */
};


////////////////////
// Number protocol
////////////////////

__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < numEls; i += numThreads) {
        dest[i] = a[i] + b[i];
    }
}
static PyObject *
CudaNdarray_add(PyObject* py_self, PyObject * py_other)
{
    if (! CudaNdarray_Check(py_self)) {
        PyErr_SetString(PyExc_TypeError, "need a CudaNdarray on left");
        return NULL;
    }
    if (! CudaNdarray_Check(py_other)) {
        PyErr_SetString(PyExc_TypeError, "need a CudaNdarray on right");
        return NULL;
    }
    CudaNdarray * self = (CudaNdarray *)py_self;
    CudaNdarray * other = (CudaNdarray *)py_other;
    if(!CudaNdarray_is_c_contiguous(self) || !CudaNdarray_is_c_contiguous(other)){
        PyErr_SetString(PyExc_TypeError, "We have implementet only the c_contiguous version for now.");
        return NULL;      
    }

    //standard elemwise size checks
    if (self->nd != other->nd)
    {
        PyErr_SetString(PyExc_TypeError, "need same number of dims");
        return NULL;
    }
    //standard elemwise dim checks
    unsigned int size = 1;
    for (int i = 0; i< self->nd; ++i)
    {
        if (CudaNdarray_HOST_DIMS(self)[i] != CudaNdarray_HOST_DIMS(other)[i])
        {
            PyErr_SetString(PyExc_TypeError, "need same dimensions");
            return NULL;
        }
        size *= (unsigned int) CudaNdarray_HOST_DIMS(self)[i];
    }
    CudaNdarray * rval = (CudaNdarray *)CudaNdarray_new_null();
    if (!rval || CudaNdarray_alloc_contiguous(rval, self->nd, CudaNdarray_HOST_DIMS(self)))
    {
        Py_XDECREF(rval);
        return NULL;
    }

    int threads_per_block = std::min(size, (unsigned int)NUM_VECTOR_OP_THREADS_PER_BLOCK);
    int n_blocks = std::min(ceil_intdiv(size,(unsigned int)threads_per_block), (unsigned int)NUM_VECTOR_OP_BLOCKS);
    kAdd_contiguous<<<n_blocks,threads_per_block>>>(
            self->devdata, other->devdata, rval->devdata, size);
    CNDA_THREAD_SYNC;
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        PyErr_Format(PyExc_RuntimeError, "Cuda error: %s: %s.\n", "kAdd", hipGetErrorString(err));
        Py_DECREF(rval);
        return NULL;
    }
    return (PyObject *) rval;
}
__global__ void k_iAdd_3(const int d0, const int d1, const int d2,
        float* a, const int sA0, const int sA1, const int sA2,
        const float* b, const int sB0, const int sB1, const int sB2)
{
    for (int i0 = blockIdx.x; i0 < d0; i0 += gridDim.x)
    {
        for (int i1 = blockIdx.y; i1 < d1; i1 += gridDim.y)
        {
            for (int i2 = threadIdx.x; i2 < d2; i2 += blockDim.x)
            {
                a[i0*sA0 + i1*sA1 + i2*sA2] += b[i0*sB0 + i1*sB1 + i2*sB2];
            }
        }
    }
}
__global__ void k_iAdd_4(const int d0, const int d1, const int d2, const int d3,
			 float* a, const int sA0, const int sA1,
			 const int sA2, const int sA3,
			 const float* b, const int sB0, const int sB1,
			 const int sB2, const int sB3)
{
    for (int i0 = blockIdx.x; i0 < d0; i0 += gridDim.x)
    {
        for (int i1 = blockIdx.y; i1 < d1; i1 += gridDim.y)
        {
            for (int i2 = threadIdx.x; i2 < d2; i2 += blockDim.x)
            {
	      for (int i3 = threadIdx.y; i3 < d3; i3 += blockDim.y)
		{
		  a[i0*sA0 + i1*sA1 + i2*sA2 + i3*sA3] += b[i0*sB0 + i1*sB1 + i2*sB2 + i3*sB3];
		}
            }
        }
    }
}
/*
 * We need this inplace Add to support IncSubTensor
 */
static PyObject *
CudaNdarray_inplace_add(PyObject* py_self, PyObject * py_other)
{
    int verbose = 0;
    if (verbose) fprintf(stderr, "INPLACE ADD");
    if (! CudaNdarray_Check(py_self)) {
        PyErr_SetString(PyExc_TypeError, "need a CudaNdarray on left");
        return NULL;
    }
    if (! CudaNdarray_Check(py_other)) {
        PyErr_SetString(PyExc_TypeError, "need a CudaNdarray on right");
        return NULL;
    }
    CudaNdarray * self = (CudaNdarray *)py_self;
    CudaNdarray * other = (CudaNdarray *)py_other;

    //standard elemwise size checks
    if (self->nd != other->nd)
    {
        PyErr_SetString(PyExc_TypeError, "need same number of dims");
        return NULL;
    }
    //standard elemwise dim checks
    unsigned int size = 1;
    for (int i = 0; i< self->nd; ++i)
    {
        if ((CudaNdarray_HOST_DIMS(self)[i] != CudaNdarray_HOST_DIMS(other)[i])
            && (CudaNdarray_HOST_DIMS(other)[i] != 1))
        {
            PyErr_SetString(PyExc_TypeError, "need same dimensions (or broadcastable dimension)");
            return NULL;
        }
        size *= (unsigned int) CudaNdarray_HOST_DIMS(self)[i];
    }

    switch(self->nd)
    {
        case 1:
            {
                dim3 n_blocks(1, 1, 1);
                dim3 n_threads(
                        std::min(CudaNdarray_HOST_DIMS(self)[0], NUM_VECTOR_OP_THREADS_PER_BLOCK)
                    );
                k_iAdd_3<<<n_blocks, n_threads>>>(1,
                        1, //CudaNdarray_HOST_DIMS(self)[0],
                        CudaNdarray_HOST_DIMS(self)[0],
                        CudaNdarray_DEV_DATA(self),
                        1,
                        1, //CudaNdarray_HOST_STRIDES(self)[0],
                        CudaNdarray_HOST_STRIDES(self)[0],
                        CudaNdarray_DEV_DATA(other),
                        1,
                        1, //CudaNdarray_HOST_STRIDES(other)[0],
                        CudaNdarray_HOST_STRIDES(other)[0]);
                CNDA_THREAD_SYNC;
                hipError_t err = hipGetLastError();
                if( hipSuccess != err) 
                {
                    PyErr_Format(PyExc_RuntimeError, "Cuda error: %s: %s.\n", "k_iAdd", hipGetErrorString(err));
                    return NULL;
                }
                Py_INCREF(py_self);
                return py_self;
            }
        case 2:
            {
                dim3 n_blocks(1,
                        std::min(CudaNdarray_HOST_DIMS(self)[0], NUM_VECTOR_OP_BLOCKS)
                        );
                dim3 n_threads(
                        std::min(CudaNdarray_HOST_DIMS(self)[1], NUM_VECTOR_OP_THREADS_PER_BLOCK)
                    );
                k_iAdd_3<<<n_blocks, n_threads>>>(1,
                        CudaNdarray_HOST_DIMS(self)[0],
                        CudaNdarray_HOST_DIMS(self)[1],
                        CudaNdarray_DEV_DATA(self),
                        1,
                        CudaNdarray_HOST_STRIDES(self)[0],
                        CudaNdarray_HOST_STRIDES(self)[1],
                        CudaNdarray_DEV_DATA(other),
                        1,
                        CudaNdarray_HOST_STRIDES(other)[0],
                        CudaNdarray_HOST_STRIDES(other)[1]);
                CNDA_THREAD_SYNC;
                hipError_t err = hipGetLastError();
                if( hipSuccess != err) 
                {
                    PyErr_Format(PyExc_RuntimeError, "Cuda error: %s: %s.\n", "k_iAdd", hipGetErrorString(err));
                    return NULL;
                }
                Py_INCREF(py_self);
                return py_self;
            }
        case 3:
            {
                dim3 n_blocks(
                        std::min(CudaNdarray_HOST_DIMS(self)[0], NUM_VECTOR_OP_BLOCKS),
                        CudaNdarray_HOST_DIMS(self)[1]
                        );
                while (n_blocks.x * n_blocks.y > NUM_VECTOR_OP_BLOCKS) n_blocks.y /= 2;
                dim3 n_threads(
                        std::min(CudaNdarray_HOST_DIMS(self)[2], NUM_VECTOR_OP_THREADS_PER_BLOCK)
                    );
                k_iAdd_3<<<n_blocks, n_threads>>>(
                        CudaNdarray_HOST_DIMS(self)[0],
                        CudaNdarray_HOST_DIMS(self)[1],
                        CudaNdarray_HOST_DIMS(self)[2],
                        CudaNdarray_DEV_DATA(self),
                        CudaNdarray_HOST_STRIDES(self)[0],
                        CudaNdarray_HOST_STRIDES(self)[1],
                        CudaNdarray_HOST_STRIDES(self)[2],
                        CudaNdarray_DEV_DATA(other),
                        CudaNdarray_HOST_STRIDES(other)[0],
                        CudaNdarray_HOST_STRIDES(other)[1],
                        CudaNdarray_HOST_STRIDES(other)[2]);
                CNDA_THREAD_SYNC;
                hipError_t err = hipGetLastError();
                if( hipSuccess != err) 
                {
                    PyErr_Format(PyExc_RuntimeError, "Cuda error: %s: %s.\n", "k_iAdd", hipGetErrorString(err));
                    return NULL;
                }
                Py_INCREF(py_self);
                return py_self;
            }
        case 4:
            {
                dim3 n_blocks(
                        std::min(CudaNdarray_HOST_DIMS(self)[0], NUM_VECTOR_OP_BLOCKS),
                        CudaNdarray_HOST_DIMS(self)[1]
                        );
                while (n_blocks.x * n_blocks.y > NUM_VECTOR_OP_BLOCKS) n_blocks.y /= 2;
                dim3 n_threads(
                        std::min(CudaNdarray_HOST_DIMS(self)[2], NUM_VECTOR_OP_THREADS_PER_BLOCK)
                    );
                k_iAdd_4<<<n_blocks, n_threads>>>(
                        CudaNdarray_HOST_DIMS(self)[0],
                        CudaNdarray_HOST_DIMS(self)[1],
                        CudaNdarray_HOST_DIMS(self)[2],
                        CudaNdarray_HOST_DIMS(self)[3],
                        CudaNdarray_DEV_DATA(self),
                        CudaNdarray_HOST_STRIDES(self)[0],
                        CudaNdarray_HOST_STRIDES(self)[1],
                        CudaNdarray_HOST_STRIDES(self)[2],
                        CudaNdarray_HOST_STRIDES(self)[3],
                        CudaNdarray_DEV_DATA(other),
                        CudaNdarray_HOST_STRIDES(other)[0],
                        CudaNdarray_HOST_STRIDES(other)[1],
                        CudaNdarray_HOST_STRIDES(other)[2],
                        CudaNdarray_HOST_STRIDES(other)[3]);
                CNDA_THREAD_SYNC;
                hipError_t err = hipGetLastError();
                if( hipSuccess != err) 
                {
                    PyErr_Format(PyExc_RuntimeError, "Cuda error: %s: %s.\n", "k_iAdd", hipGetErrorString(err));
                    return NULL;
                }
                Py_INCREF(py_self);
                return py_self;
            }
    }

    PyErr_Format(PyExc_NotImplementedError, "inplace_add w nd=%i\n", self->nd);
    return NULL;
}

static PyNumberMethods CudaNdarrayNumberMethods =
{
     (binaryfunc)CudaNdarray_add,  //binaryfunc nb_add;
     0,  //binaryfunc nb_subtract;
     0,  //binaryfunc nb_multiply;
     0,  //binaryfunc nb_divide;
     0,  //binaryfunc nb_remainder;
     0,  //binaryfunc nb_divmod;
     0,  //ternaryfunc nb_power;
     0,  //unaryfunc nb_negative;
     0,  //unaryfunc nb_positive;
     0,  //unaryfunc nb_absolute;
     0,  //inquiry nb_nonzero;       /* Used by PyObject_IsTrue */
     0,  //unaryfunc nb_invert;
     0,  //binaryfunc nb_lshift;
     0,  //binaryfunc nb_rshift;
     0,  //binaryfunc nb_and;
     0,  //binaryfunc nb_xor;
     0,  //binaryfunc nb_or;
     0,  //coercion nb_coerce;       /* Used by the coerce() function */
     0,  //unaryfunc nb_int;
     0,  //unaryfunc nb_long;
     0,  //unaryfunc nb_float;
     0,  //unaryfunc nb_oct;
     0,  //unaryfunc nb_hex;

     /* Added in release 2.0 */
     (binaryfunc)CudaNdarray_inplace_add,  //binaryfunc nb_inplace_add;
     0,  //binaryfunc nb_inplace_subtract;
     0,  //binaryfunc nb_inplace_multiply;
     0,  //binaryfunc nb_inplace_divide;
     0,  //binaryfunc nb_inplace_remainder;
     0,  //ternaryfunc nb_inplace_power;
     0,  //binaryfunc nb_inplace_lshift;
     0,  //binaryfunc nb_inplace_rshift;
     0,  //binaryfunc nb_inplace_and;
     0,  //binaryfunc nb_inplace_xor;
     0,  //binaryfunc nb_inplace_or;

     /* Added in release 2.2 */
     0,  //binaryfunc nb_floor_divide;
     0,  //binaryfunc nb_true_divide;
     0,  //binaryfunc nb_inplace_floor_divide;
     0,  //binaryfunc nb_inplace_true_divide;

#if PY_MINOR_VERSION > 4
     /* Added in release 2.5 */
     0  //unaryfunc nb_index;
#endif
};


/////////////////////
// Mapping protocol
/////////////////////

static Py_ssize_t 
CudaNdarray_len(PyObject * py_self)
{
    CudaNdarray * self = (CudaNdarray*) py_self;
    if (self->nd <= 0)
    {
        return (Py_ssize_t) 0;
    }
    else
    {
        return (Py_ssize_t) CudaNdarray_HOST_DIMS(self)[0];
    }
}

static PyObject *
CudaNdarray_Subscript(PyObject * py_self, PyObject * key)
{
    int verbose = 0;
    if (verbose) fprintf(stderr, "Subscript .... \n");
    CudaNdarray * self = (CudaNdarray*) py_self;
    PyObject * py_rval = NULL;
    CudaNdarray * rval = NULL;

    if (key == Py_Ellipsis)
    {
        Py_INCREF(py_self);
        return py_self;
    }
    else if (PyInt_Check(key)) //INDEXING BY INTEGER
    {
        if (self->nd == 0)
        {
            PyErr_SetString(PyExc_NotImplementedError, "index into 0-d array");
            return NULL;
        }

        int d_idx = PyInt_AsLong(key);
        int d_dim = CudaNdarray_HOST_DIMS(self)[0];
        int offset = 0;

        if ((d_idx >= 0) && (d_idx < d_dim))
        {
            //normal indexing
            offset += d_idx * CudaNdarray_HOST_STRIDES(self)[0];
        }
        else if ((d_idx < 0) && (d_idx >= -d_dim))
        {
            //end-based indexing
            offset += (d_dim - d_idx) * CudaNdarray_HOST_STRIDES(self)[0];
        }
        else
        {
            PyErr_SetString(PyExc_IndexError, "index out of bounds");
            Py_DECREF(rval);
            return NULL;
        }

        //allocate our subtensor view
        py_rval = CudaNdarray_new_nd(self->nd - 1);
        rval = (CudaNdarray*) py_rval;
        if (!rval) return NULL;
        assert (0 == rval->data_allocated);

        //initialize the view's data pointer to our own.
        if (CudaNdarray_set_device_data(rval, CudaNdarray_DEV_DATA(self) + offset, self))
        {
            Py_DECREF(rval);
            return NULL;
        }
        for (int d = 1; d < self->nd; ++d)
        {
            CudaNdarray_set_stride(rval, d-1, CudaNdarray_HOST_STRIDES(self)[d]);
            CudaNdarray_set_dim(rval, d-1, CudaNdarray_HOST_DIMS(self)[d]);
        }
    }
    else if (PySlice_Check(key)) //INDEXING BY SLICE
    {
        if (self->nd == 0)
        {
            PyErr_SetString(PyExc_NotImplementedError, "index into 0-d array");
            return NULL;
        }

        int d_dim = CudaNdarray_HOST_DIMS(self)[0];
        Py_ssize_t start, stop, step, slen;
        if (PySlice_GetIndicesEx((PySliceObject*)key, d_dim, &start, &stop, &step, &slen))
        {
            return NULL;
        }
        if (verbose)
        {
            std::cerr << "start " << start << "\n";
            std::cerr << "stop " << stop << "\n";
            std::cerr << "step " << step << "\n";
            std::cerr << "slen " << slen << "\n";
        }

        //allocate our subtensor view
        py_rval = CudaNdarray_new_nd(self->nd);
        rval = (CudaNdarray*) py_rval;
        if (!rval) return NULL;
        assert (0 == rval->data_allocated);


        //initialize the view's data pointer to our own.
        if (CudaNdarray_set_device_data(rval,
                    CudaNdarray_DEV_DATA(self) + start * CudaNdarray_HOST_STRIDES(self)[0],
                    self))
        {
            Py_DECREF(rval);
            return NULL;
        }
        //initialize dimension 0 of rval
        CudaNdarray_set_stride(rval, 0, step * CudaNdarray_HOST_STRIDES(self)[0]);
        CudaNdarray_set_dim(rval, 0, slen);
        if (verbose) std::cerr << "rval stride " << CudaNdarray_HOST_STRIDES(rval)[0] << "\n";
        // initialize dimensions > 0 of rval
        for (int d = 1; d < self->nd; ++d)
        {
            CudaNdarray_set_stride(rval, d, CudaNdarray_HOST_STRIDES(self)[d]);
            CudaNdarray_set_dim(rval, d, CudaNdarray_HOST_DIMS(self)[d]);
        }
    }
    else if (PyTuple_Check(key)) //INDEXING BY TUPLE
    {
        //elements of the tuple can be either integers or slices
        //the dimensionality of the view we will return is diminished for each slice in the tuple
        
        if (PyTuple_Size(key) > self->nd)
        {
            PyErr_SetString(PyExc_IndexError, "index error");
            return NULL;
        }

        //calculate the number of dimensions in the return value
        int rval_nd = self->nd;
        for (int d = 0; d < PyTuple_Size(key); ++d)
        {
            rval_nd -= PyInt_Check(PyTuple_GetItem(key, d));
        }

        //allocate our subtensor view
        py_rval = CudaNdarray_new_nd(rval_nd);
        rval = (CudaNdarray*) py_rval;
        if (!rval) return NULL;
        assert (0 == rval->data_allocated);

        //initialize the view's data pointer to our own.
        if (CudaNdarray_set_device_data(rval, CudaNdarray_DEV_DATA(self), self))
        {
            Py_DECREF(rval);
            return NULL;
        }

        // rval_d will refer to the current dimension in the rval.  
        // It will not be incremented for integer keys, but will be incremented for slice
        // keys
        int rval_d = 0;

        for (int d = 0; d < self->nd; ++d)
        {
            // keys can be shorter than self->nd.
            // when that happens, it means that the remaining dimensions are "full slices"
            if (d >=PyTuple_Size(key)) 
            {
                CudaNdarray_set_stride(rval, rval_d, CudaNdarray_HOST_STRIDES(self)[d]);
                CudaNdarray_set_dim(rval, rval_d, CudaNdarray_HOST_DIMS(self)[d]);
                ++rval_d;
            }
            else
            {
                PyObject * key_d = PyTuple_GetItem(key, d);

                if (PySlice_Check(key_d))
                {
                    Py_ssize_t start, stop, step, slen;
                    if (PySlice_GetIndicesEx((PySliceObject*)key_d, CudaNdarray_HOST_DIMS(self)[d], &start, &stop, &step, &slen))
                    {
                        Py_DECREF(rval);
                        return NULL;
                    }
                    rval->devdata += start * CudaNdarray_HOST_STRIDES(self)[d];
                    CudaNdarray_set_stride(rval, rval_d, step * CudaNdarray_HOST_STRIDES(self)[d]);
                    CudaNdarray_set_dim(rval, rval_d, slen);
                    if (0)
                    {
                        std::cerr << "start " << start << "\n";
                        std::cerr << "stop " << stop << "\n";
                        std::cerr << "step " << step << "\n";
                        std::cerr << "slen " << slen << "\n";
                    }
                    ++rval_d;
                }
                else if (PyInt_Check(key_d))
                {
                    int d_idx = PyInt_AsLong(key_d);
                    int d_dim = CudaNdarray_HOST_DIMS(self)[d];

                    if ((d_idx >= 0) && (d_idx < d_dim))
                    {
                        //normal indexing
                        rval->devdata += d_idx * CudaNdarray_HOST_STRIDES(self)[d];
                    }
                    else if ((d_idx < 0) && (d_idx >= -d_dim))
                    {
                        //end-based indexing
                        rval->devdata += (d_dim + d_idx) * CudaNdarray_HOST_STRIDES(self)[d];
                    }
                    else
                    {
                        PyErr_SetString(PyExc_IndexError, "index out of bounds");
                        Py_DECREF(rval);
                        return NULL;
                    }
                }
                else
                {
                    PyErr_SetString(PyExc_IndexError, "index must be either int or slice");
                    Py_DECREF(rval);
                    return NULL;
                }
            }
        }
    }
    else
    {
        PyErr_SetString(PyExc_NotImplementedError, "Unknown key type");
        return NULL;
    }
    if (py_rval)
    {
        if (verbose) fprint_CudaNdarray(stderr, self);
        if (verbose) fprint_CudaNdarray(stderr, rval);
    }
    return py_rval;
}

PyMappingMethods CudaNdarrayMappingMethods = {
    CudaNdarray_len, //lenfunc mp_length;
    CudaNdarray_Subscript, //binaryfunc mp_subscript;
    0, //objobjargproc mp_ass_subscript;
};

////////////////////
//
////////////////////

static PyObject *
CudaNdarray_get_shape(CudaNdarray *self, void *closure)
{
    if (self->nd < 0)
    {
        PyErr_SetString(PyExc_ValueError, "CudaNdarray not initialized");
        return NULL;
    }
    PyObject * rval = PyTuple_New(self->nd);
    for (int i = 0; i < self->nd; ++i)
    {
        if (!rval || PyTuple_SetItem(rval, i, PyInt_FromLong(CudaNdarray_HOST_DIMS(self)[i])))
        {
            Py_XDECREF(rval);
            return NULL;
        }

    }
    return rval;
}

static int
CudaNdarray_set_shape(CudaNdarray *self, PyObject *value, void *closure)
{
    PyErr_SetString(PyExc_NotImplementedError, "TODO: call reshape");
    return -1;
}

static PyObject *
CudaNdarray_get_strides(CudaNdarray *self, void *closure)
{
    if (self->nd < 0)
    {
        PyErr_SetString(PyExc_ValueError, "CudaNdarray not initialized");
        return NULL;
    }
    PyObject * rval = PyTuple_New(self->nd);
    for (int i = 0; i < self->nd; ++i)
    {
        if (!rval || PyTuple_SetItem(rval, i, PyInt_FromLong(CudaNdarray_HOST_STRIDES(self)[i])))
        {
            Py_XDECREF(rval);
            return NULL;
        }

    }
    return rval;
}

static int
CudaNdarray_set_strides(CudaNdarray *self, PyObject *value, void *closure)
{
    PyErr_SetString(PyExc_NotImplementedError, "");
    return -1;
}

static PyObject *
CudaNdarray_get_dev_data(CudaNdarray *self, void *closure)
{
    float * p =  CudaNdarray_DEV_DATA(self);
    //printf("get_dev_data %p %li \n", p, (long int)p );
    return PyInt_FromLong((long int) CudaNdarray_DEV_DATA(self));
}

static int
CudaNdarray_set_dev_data(CudaNdarray *self, PyObject *value, void *closure)
{
    long int newdevdata = PyInt_AsLong(value);
    //printf("set_dev_data %p %li \n",(float*)newdevdata ,newdevdata);
    if (PyErr_Occurred())
    {
        return -1;
    }
    return  CudaNdarray_set_device_data(self, (float*)newdevdata, (CudaNdarray*)self->base);
}

static PyGetSetDef CudaNdarray_getset[] = {
    {"shape", 
        (getter)CudaNdarray_get_shape, 
        (setter)CudaNdarray_set_shape, 
        "shape of this ndarray (tuple)",
        NULL},
    {"_strides", 
        (getter)CudaNdarray_get_strides, 
        (setter)CudaNdarray_set_strides, 
        "data pointer strides (in elements)",
        NULL},
    {"_dev_data", 
        (getter)CudaNdarray_get_dev_data, 
        (setter)CudaNdarray_set_dev_data,
        "device data pointer",
        NULL},
    {NULL, NULL, NULL, NULL}  /* Sentinel */
};



static PyTypeObject CudaNdarrayType = 
{
    PyObject_HEAD_INIT(NULL)
    0,                         /*ob_size*/
    "noddy.CudaNdarray",       /*tp_name*/
    sizeof(CudaNdarray),       /*tp_basicsize*/
    0,                         /*tp_itemsize*/
    (destructor)CudaNdarray_dealloc, /*tp_dealloc*/
    0,                         /*tp_print*/
    0,                         /*tp_getattr*/
    0,                         /*tp_setattr*/
    0,                         /*tp_compare*/
    0,                         /*tp_repr*/
    &CudaNdarrayNumberMethods, /*tp_as_number*/
    0,                         /*tp_as_sequence*/
    &CudaNdarrayMappingMethods,/*tp_as_mapping*/
    0,                         /*tp_hash */
    0,                         /*tp_call*/
    0,                         /*tp_str*/
    0,                         /*tp_getattro*/
    0,                         /*tp_setattro*/
    0,                         /*tp_as_buffer*/
    Py_TPFLAGS_DEFAULT | Py_TPFLAGS_BASETYPE | Py_TPFLAGS_CHECKTYPES, /*tp_flags*/
    "CudaNdarray objects",     /* tp_doc */
    0,                         /* tp_traverse */
    0,                         /* tp_clear */
    0,                         /* tp_richcompare */
    0,                         /* tp_weaklistoffset */
    0,                         /* tp_iter */
    0,                         /* tp_iternext */
    CudaNdarray_methods,       /* tp_methods */
    CudaNdarray_members,       /* tp_members */
    CudaNdarray_getset,        /* tp_getset */
    0,                         /* tp_base */
    0,                         /* tp_dict */
    0,                         /* tp_descr_get */
    0,                         /* tp_descr_set */
    0,                         /* tp_dictoffset */
    (initproc)CudaNdarray_init,/* tp_init */
    0,                         /* tp_alloc */
    CudaNdarray_new,           /* tp_new */
};

PyObject *
CudaNdarray_gpu_init(PyObject* _unsed, PyObject * args)
{
  int card_nb=0;

  if (! PyArg_ParseTuple(args, "|i", &card_nb))
    return NULL; 

  int deviceCount;

  hipError_t err = hipGetDeviceCount(&deviceCount);
  if( hipSuccess != err) {
    //TODO: put this as a warning and let theano continue on the cpu...
    PyErr_Format(PyExc_RuntimeError, "ERROR: Not able to get the number of gpu available.");
    return NULL;
  }
  if (deviceCount <= 0) {
    //TODO: put this as a warning and let theano continue on the cpu...
    PyErr_Format(PyExc_RuntimeError, "ERROR: Can't use the GPU, no devices supporting CUDA.\n");
    return NULL;
  }
  if(card_nb<0 || card_nb>(deviceCount-1)){
    PyErr_Format(PyExc_RuntimeError, "ERROR: bad device number %d. Their is only %d device available\n",
		 card_nb, deviceCount);
    return NULL;
  }

  hipDeviceProp_t deviceProp;
  err=hipGetDeviceProperties(&deviceProp, card_nb);
  if( hipSuccess != err) {
    PyErr_Format(PyExc_RuntimeError, "ERROR: Was not able to get the property of the gpu %i.",
		 card_nb);
    exit(-1);
  }

  if(deviceProp.major == 9999 && deviceProp.minor == 9999 ){
    PyErr_Format(PyExc_RuntimeError, "WARNING: Their is no device that support CUDA.\n");
    return NULL;    
  }
  
  fprintf(stderr, "Using gpu device %d: %s\n", card_nb, deviceProp.name);

  err = hipSetDevice(card_nb);
  if( hipSuccess != err) {
    PyErr_Format(PyExc_RuntimeError, "ERROR: Was not able to set the device. %s\n", hipGetErrorString(err));
    return NULL;
  }

  Py_INCREF(Py_None);
  return Py_None;
}

PyObject *
CudaNdarray_Dot(PyObject* _unsed, PyObject * args)
{
    PyObject *l=NULL;
    PyObject *r=NULL;
    PyObject * rval = NULL;

    if (! PyArg_ParseTuple(args, "OO", &l, &r))
        return NULL; 

    if (!CudaNdarray_Check(l) || !CudaNdarray_Check(r))
    {
        PyErr_SetString(PyExc_TypeError, "CudaNdarray arguments required ");
        goto CudaNdarray_dot_fail;
    }
    if (((CudaNdarray*)l)->nd != 2)
    {
        PyErr_SetString(PyExc_TypeError, "need 2d CudaNdarray arg for now");
        goto CudaNdarray_dot_fail;
    }
    if (((CudaNdarray*)r)->nd != 2)
    {
        PyErr_SetString(PyExc_TypeError, "need 2d CudaNdarray arg for now");
        goto CudaNdarray_dot_fail;
    }
    rval = CudaNdarray_new_null();
    if (!rval)
    {
        goto CudaNdarray_dot_fail;
    }
    int dims[2];
    dims[0] = CudaNdarray_HOST_DIMS((CudaNdarray*)l)[0];
    dims[1] = CudaNdarray_HOST_DIMS((CudaNdarray*)r)[1];
    if (CudaNdarray_alloc_contiguous((CudaNdarray*)rval, 2, dims))
    {
        goto CudaNdarray_dot_fail;
    }
    if (CudaNdarray_gemm(1.0, (CudaNdarray*)l, (CudaNdarray*)r, 0.0, (CudaNdarray*)rval))
    {
        goto CudaNdarray_dot_fail;
    }

    return rval;

    CudaNdarray_dot_fail:
    Py_XDECREF(rval);
    return NULL;
}
static PyObject *
CudaNdarray_Conv_VARARGS(PyObject * _unused, PyObject *args, PyObject * kwargs)
{
    //Mandatory args
    PyObject *img = NULL;
    PyObject *kern = NULL;
    PyObject *mode_str = NULL;

    //Optional args
    PyObject *out = NULL;
    PyObject *subsample = NULL;
    PyObject *logical_img_shape = NULL;
    PyObject *logical_kern_shape = NULL;
    PyObject *kern_align = NULL;
    int version = -1;
    int verbose = 0;
    // the output_downsampling arguments as integers
    const int od_0_orig = 1;
    const int od_1_orig = 1;
    int od_0 = od_0_orig;
    int od_1 = od_1_orig;

    PyObject *out_2 = NULL;

    static char *kwlist[] = {"img", "kern", "mode", "out", "subsample", "logical_img_shape", "logical_kern_shape", "kern_align", "version", "verbose", NULL };

    if (! PyArg_ParseTupleAndKeywords(args, kwargs, "OOS|OOOOOii", kwlist,
                &img, &kern, &mode_str, 
                &out, &subsample, &logical_img_shape, &logical_kern_shape, &kern_align, &version, &verbose))
        return NULL; 
    int mode;
    if (strcmp(PyString_AsString(mode_str), "full") == 0)
    {
        mode = ConvMode_FULL;
    }
    else if (strcmp(PyString_AsString(mode_str), "valid") == 0)
    {
        mode = ConvMode_VALID;
    }
    else
    {
        PyErr_SetString(PyExc_ValueError, "mode must be one of 'full' or 'valid'");
        return NULL;
    }
    if (!CudaNdarray_Check(img))
    {
        PyErr_SetString(PyExc_TypeError, "img argument must be a CudaNdarray");
        return NULL;
    }
    if (!CudaNdarray_Check(kern))
    {
        PyErr_SetString(PyExc_TypeError, "kern argument must be a CudaNdarray");
        return NULL;
    }
    if (out && CudaNdarray_Check(out))
    {
        out_2 = out;
    }
    else if (out && Py_None != out)
    {
        fprintf(stderr, "Warning: Conv is ignoring 'out' argument that wasn't a CudaNdarray.\n");
    }
    if (subsample)
    {
        if ((!PySequence_Check(subsample))
                || (PySequence_Length(subsample) != 2))
        {
            PyErr_SetString(PyExc_TypeError, "'subsample' argument must be a length-2 sequence of integers");
            return NULL;
        }
        PyObject *py_od_0 = PySequence_GetItem(subsample, 0);
        PyObject *py_od_1 = PySequence_GetItem(subsample, 1);
        od_0 = PyInt_AsLong(py_od_0);
        od_1 = PyInt_AsLong(py_od_1);
        if (PyErr_Occurred())
        {
            od_0 = od_0_orig;
            od_1 = od_1_orig;
            Py_XDECREF(py_od_0);
            Py_XDECREF(py_od_1);
            PyErr_SetString(PyExc_TypeError, "'subsample' argument must be a length-2 sequence of integers");
            return NULL;
        }
    }
    return CudaNdarray_Conv((CudaNdarray*)img, (CudaNdarray*)kern, (CudaNdarray*)out_2, mode, od_0, od_1, version, verbose);
}

static PyObject * 
filter(PyObject* __unsed_self, PyObject *args) // args = (data, broadcastable, strict)
{
    PyObject *py_data=NULL;
    PyArrayObject * data = NULL;
    int strict = 0;
    PyObject * broadcastable=NULL;

    if (!PyArg_ParseTuple(args, "OOi", &py_data, &broadcastable, &strict)) return NULL;

    if (!PyTuple_Check(broadcastable)){
        PyErr_SetString(PyExc_TypeError, "broadcastable arg should be a tuple of int.");
        return NULL;
    }
    Py_INCREF(py_data);
    Py_INCREF(broadcastable);

    CudaNdarray * cnda = (CudaNdarray*)py_data;

    if (strict or CudaNdarray_Check(py_data))
    {
        //TODO: support non-strict "casting" from a vt to the broadcastable/type/size that we need.
        if (!CudaNdarray_Check(py_data)) 
        {
            Py_DECREF(py_data);
            Py_DECREF(broadcastable);
            PyErr_SetString(PyExc_TypeError, "strict mode requires CudaNdarray");
            return NULL;
        }
        if (cnda->nd != PyTuple_Size(broadcastable))
        {
            Py_DECREF(py_data);
            Py_DECREF(broadcastable);
            PyErr_Format(PyExc_TypeError, "Wrong rank: %i vs %li", cnda->nd, (long)PyTuple_Size(broadcastable));
            return NULL;
        }
        for (int i = 0; i < cnda->nd; ++i)
        {
            if ((CudaNdarray_HOST_DIMS(cnda)[i] > 1) and PyInt_AsLong(PyTuple_GetItem(broadcastable, Py_ssize_t(i))))
            {
                PyErr_Format(PyExc_TypeError, "Non-unit size in broadcastable vt dimension %i", i);
                Py_DECREF(py_data);
                Py_DECREF(broadcastable);
                return NULL;
            }
        }
        Py_DECREF(broadcastable);
        return py_data;
    }
    else
    {
        data = (PyArrayObject*)PyArray_FromObject(py_data, REAL_TYPENUM, PyTuple_Size(broadcastable), PyTuple_Size(broadcastable));
        if (!data)
        {
            //err message already defined
            Py_DECREF(py_data);
            Py_DECREF(broadcastable);
            return NULL;
        }
        for (int i = 0; i < data->nd; ++i)
        {
            if ((data->dimensions[i] > 1) and PyInt_AsLong(PyTuple_GetItem(broadcastable, Py_ssize_t(i))))
            {
                PyErr_Format(PyExc_TypeError, "Non-unit size in broadcastable dimension %i", i);
                Py_DECREF(data);
                Py_DECREF(py_data);
                Py_DECREF(broadcastable);
                return NULL;
            }
        }
        CudaNdarray * rval = (CudaNdarray*) CudaNdarray_new_null();
        if (CudaNdarray_CopyFromArray(rval, data))
        {
            Py_DECREF(rval);
            rval = NULL;
        }
        Py_DECREF(data);
        Py_DECREF(py_data);
        Py_DECREF(broadcastable);
        return (PyObject*)rval;
    }
}

static PyMethodDef module_methods[] = {
    {"dot", CudaNdarray_Dot, METH_VARARGS, "Returns the matrix product of two CudaNdarray arguments."},
    {"conv", (PyCFunction)CudaNdarray_Conv_VARARGS, METH_VARARGS|METH_KEYWORDS, "Returns the 2D convolution of one CudaNdarray argument with another. WRITEME"},
    {"gpu_init", CudaNdarray_gpu_init, METH_VARARGS, "Allow to select the gpu card to use."},
    {"filter", filter, METH_VARARGS, "no doc!"},    
    {NULL, NULL, NULL, NULL}  /* Sentinel */
};

#ifndef PyMODINIT_FUNC  /* declarations for DLL import/export */
#define PyMODINIT_FUNC void
#endif
PyMODINIT_FUNC
initcuda_ndarray(void) 
{
    import_array();

    PyObject* m;

    if (PyType_Ready(&CudaNdarrayType) < 0)
        return;

    m = Py_InitModule3("cuda_ndarray", module_methods,
                       "Example module that creates an extension type.");

    if (m == NULL)
        return;

    Py_INCREF(&CudaNdarrayType);
    PyModule_AddObject(m, "CudaNdarray", (PyObject *)&CudaNdarrayType);

    //    cublasInit();
    if (0&&HIPBLAS_STATUS_SUCCESS != cublasGetError())
    {
        std::cerr << "WARNING: initcuda_ndarray: error initializing device\n";
    }
    if (0) //TODO: is this necessary?
    {
        int deviceId = 0; // TODO: what number goes here?
        hipSetDevice(deviceId);
        hipError_t err = hipGetLastError();
        if( hipSuccess != err) 
        {
            std::cerr << "Error in SetDevice:" << hipGetErrorString(err) << "\n";
        }                         
    }
}


//////////////////////////////////////
//
// C API FOR CudaNdarray
//
//////////////////////////////////////

int 
CudaNdarray_Check(const PyObject * ob)
{
    //TODO: doesn't work with inheritance
    return CudaNdarray_CheckExact(ob);
}
int 
CudaNdarray_CheckExact(const PyObject * ob)
{
    return ((ob->ob_type == &CudaNdarrayType) ? 1 : 0);
}

PyObject * 
CudaNdarray_New(int nd)
{
    CudaNdarray *self = (CudaNdarray *)CudaNdarrayType.tp_alloc(&CudaNdarrayType, 0);
    if (self == NULL) 
    {
        PyErr_SetString(PyExc_RuntimeError, "CudaNdarray_new_null failed to allocate self");
        return NULL;
    }
    CudaNdarray_null_init(self);
    
    if (nd == 0)
    {
        self->nd = 0;
    }
    else if (nd > 0)
    {
        if (CudaNdarray_set_nd(self, nd))
        {
            Py_DECREF(self);
            return NULL;
        }
    }
    return (PyObject *)self;
}



//////////////////////////////
//
// Published helper functions
//
//////////////////////////////

int 
cublas_init() 
{
    cublasInit();
    if (HIPBLAS_STATUS_SUCCESS != cublasGetError())
    {
        PyErr_SetString(PyExc_RuntimeError, "error initializing device");
        return -1;
    }
    return 0;
}
int 
cublas_shutdown() 
{
    cublasShutdown();
    if (HIPBLAS_STATUS_SUCCESS != cublasGetError())
    {
        PyErr_SetString(PyExc_RuntimeError, "error shutting down device");
        return -1;
    }
    return 0;
}

int 
CudaNdarray_CopyFromArray(CudaNdarray * self, PyArrayObject*obj)
{
    int err = CudaNdarray_alloc_contiguous(self, obj->nd, obj->dimensions);
    if (err) {
        return err;
    }

    int typenum = PyArray_TYPE(obj);
    if (typenum != REAL_TYPENUM)
    {
        PyErr_SetString(PyExc_TypeError, "can only copy from float arrays");
        return -1;
    }
    assert( 4 ==  PyArray_ITEMSIZE(obj));
    PyObject * py_src = PyArray_ContiguousFromAny((PyObject*)obj, typenum, self->nd, self->nd);
    if (!py_src) {
        return -1;
    }
    hipblasSetVector(PyArray_SIZE(py_src),
            sizeof(real), 
            PyArray_DATA(py_src), 1,
            self->devdata, 1);
    CNDA_THREAD_SYNC;
    if (HIPBLAS_STATUS_SUCCESS != cublasGetError())
    {
        PyErr_SetString(PyExc_RuntimeError, "error copying data to device memory");
        Py_DECREF(py_src);
        return -1;
    }
    Py_DECREF(py_src);
    return 0;
}
bool 
CudaNdarray_is_c_contiguous(const CudaNdarray * self)
{
    bool c_contiguous = true;
    int size = 1;
    for (int i = self->nd-1; (i >= 0) and c_contiguous; --i)
    {
        if (CudaNdarray_HOST_DIMS(self)[i] == 1)
            continue;
        //std::cerr << i << " "<< str << "BBBB\n";
        if (CudaNdarray_HOST_STRIDES(self)[i] != size)
        {
            c_contiguous = false;
        }
        size = size * CudaNdarray_HOST_DIMS(self)[i];
    }
    return c_contiguous;
}
PyObject *
CudaNdarray_new_null()
{
    //TODO: this function is deprecated... do not use. Consider removing.
    return CudaNdarray_New(-1);
}
PyObject *
CudaNdarray_new_nd(int nd)
{
    CudaNdarray * rval = (CudaNdarray*) CudaNdarray_new_null();
    if (!rval || CudaNdarray_set_nd(rval, nd))
    {
        Py_XDECREF(rval);
        rval = NULL;
    }
    return (PyObject *) rval;
}

int CudaNdarray_set_device_data(CudaNdarray * self, float * data, CudaNdarray * base)
{
    if (self->data_allocated)
    {
        assert(self->devdata);
        cublasFree(self->devdata);
        if (HIPBLAS_STATUS_SUCCESS != cublasGetError())
        {
            PyErr_SetString(PyExc_MemoryError, "error freeing device memory");
            self->devdata = NULL;
            self->data_allocated = 0;
            return -1;
        }
    }
    //N.B. XDECREF and XINCREF are no-ops for NULL pointers
    if (self->base != (PyObject*)base)
    {
        Py_XDECREF(self->base);
        self->base = (PyObject*)base;
        Py_XINCREF(self->base);
    }
    self->data_allocated = 0;
    self->devdata = data;
    return 0;
}

static __global__ void k_copy_1d(const int N, const float * x, const int sx, float * y, const int sy)
{
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < N; i += gridDim.x*blockDim.x)
    {
        y[i*sy] = x[i*sx];
    }
}

//copy from other into self
int CudaNdarray_CopyFromCudaNdarray(CudaNdarray * self, CudaNdarray * other)
{
    int verbose = 0;
    //standard elemwise size checks
    if (self->nd == -1)
    {
        PyErr_SetString(PyExc_TypeError, "can't copy into un-initialized CudaNdarray");
        return -1;
    }
    if (self->nd != other->nd)
    {
        PyErr_SetString(PyExc_TypeError, "need same number of dims");
        return -1;
    }
    //standard elemwise dim checks (also compute total size)
    unsigned int size = 1;
    for (int i = 0; i< self->nd; ++i)
    {
        if (CudaNdarray_HOST_DIMS(self)[i] != CudaNdarray_HOST_DIMS(other)[i])
        {
            PyErr_SetString(PyExc_TypeError, "need same dimensions");
            return -1;
        }
        size *= (unsigned int) CudaNdarray_HOST_DIMS(self)[i];
    }
    if (CudaNdarray_is_c_contiguous(self) && CudaNdarray_is_c_contiguous(other))
    {
        hipblasScopy(size, CudaNdarray_DEV_DATA(other), 1, CudaNdarray_DEV_DATA(self), 1);
        if (HIPBLAS_STATUS_SUCCESS != cublasGetError())
        {
            PyErr_SetString(PyExc_RuntimeError, "Error copying memory");
            return -1;
        }
        return 0;
    }
    //TODO: rewrite these copy operations to be more efficient
    //      See, for example the transpose example in the cuda_sdk.
    switch (self->nd)
    {
        case 0: // scalar
            {
                assert (size==1);
                hipblasScopy(1, CudaNdarray_DEV_DATA(other), 1, CudaNdarray_DEV_DATA(self), 1);
                CNDA_THREAD_SYNC;
                if (HIPBLAS_STATUS_SUCCESS != cublasGetError())
                {
                    PyErr_SetString(PyExc_RuntimeError, "Error copying memory");
                    return -1;
                }
            }; break;
        case 1: // vector
            {
                if (verbose) fprintf(stderr, "Copying non-contiguous vector\n");
                if (verbose) fprint_CudaNdarray(stderr, other);
                unsigned int n_blocks = std::min(size, (unsigned int)NUM_VECTOR_OP_BLOCKS);
                unsigned int n_threads = std::min(ceil_intdiv(size, n_blocks), (unsigned int)NUM_VECTOR_OP_THREADS_PER_BLOCK);
                k_copy_1d<<<n_blocks, n_threads>>>(size,
                        CudaNdarray_DEV_DATA(other), CudaNdarray_HOST_STRIDES(other)[0],
                        CudaNdarray_DEV_DATA(self), CudaNdarray_HOST_STRIDES(self)[0]);
                CNDA_THREAD_SYNC;
                hipError_t err = hipGetLastError();
                if( hipSuccess != err) 
                {
                    PyErr_Format(PyExc_RuntimeError, "Cuda error: %s: %s. (n_blocks=%i, n_threads_per_block=%i)\n", "k_copy_1d", hipGetErrorString(err), n_blocks, n_threads);
                    return -1;
                }                         
            }; break;
        default:
            {
                assert (hipSuccess == hipGetLastError());
                // call worker routine
                unsigned int n_blocks = std::min(size, (unsigned int)NUM_VECTOR_OP_BLOCKS);
                unsigned int threads_per_block = std::min(ceil_intdiv(size, n_blocks), (unsigned int)NUM_VECTOR_OP_THREADS_PER_BLOCK);
                //copy from other into self
                k_elemwise_unary_rowmajor_copy<<<n_blocks, threads_per_block>>>(
                        size, 
                        (unsigned int)other->nd,
                        (const int *)CudaNdarray_DEV_DIMS(other),
                        (const float*)CudaNdarray_DEV_DATA(other), (const int *)CudaNdarray_DEV_STRIDES(other),
                        CudaNdarray_DEV_DATA(self),  (const int *)CudaNdarray_DEV_STRIDES(self));
                CNDA_THREAD_SYNC;
                hipError_t err = hipGetLastError();
                if( hipSuccess != err) 
                {
                    //fprint_CudaNdarray(stderr, self);
                    //fprint_CudaNdarray(stderr, other);
                    PyErr_Format(PyExc_RuntimeError, "Cuda error: %s: %s. (n_blocks=%i, n_threads_per_block=%i)\n", "k_elemwise_unary_rowmajor_copy", hipGetErrorString(err), n_blocks, threads_per_block);
                    return -1;
                }                         
            }
    };
    return 0;
}

int CudaNdarray_gemm(float alpha, const CudaNdarray * A, const CudaNdarray * B, float beta, CudaNdarray * C)
{
    if (A->nd != 2) { PyErr_SetString(PyExc_ValueError, "non-matrix arg to gemm"); return -1; }
    if (B->nd != 2) { PyErr_SetString(PyExc_ValueError, "non-matrix arg to gemm"); return -1; }
    if (C->nd != 2) { PyErr_SetString(PyExc_ValueError, "non-matrix arg to gemm"); return -1; }

    if ((CudaNdarray_HOST_DIMS(A)[1] != CudaNdarray_HOST_DIMS(B)[0])
            || (CudaNdarray_HOST_DIMS(A)[0] != CudaNdarray_HOST_DIMS(C)[0])
            || (CudaNdarray_HOST_DIMS(B)[1] != CudaNdarray_HOST_DIMS(C)[1]))
    { 
        PyErr_Format(PyExc_ValueError, "dimension mismatch in args to gemm (%i,%i)x(%i,%i)->(%i,%i)",
                CudaNdarray_HOST_DIMS(A)[0],
                CudaNdarray_HOST_DIMS(A)[1],
                CudaNdarray_HOST_DIMS(B)[0],
                CudaNdarray_HOST_DIMS(B)[1],
                CudaNdarray_HOST_DIMS(C)[0],
                CudaNdarray_HOST_DIMS(C)[1]);
        return -1; 
    }

    // a matrix has non-unit size and non-unit stride in both directions, we can't operate in-place
    // TODO: make a copy instead of returning in error
    if (((CudaNdarray_HOST_DIMS(A)[0] > 1) && (CudaNdarray_HOST_STRIDES(A)[0] != 1)) && ((CudaNdarray_HOST_DIMS(A)[1] > 1) && (CudaNdarray_HOST_STRIDES(A)[1] != 1)))
    { PyErr_SetString(PyExc_NotImplementedError, "non-unit stride in gemm arg"); return -1; }
    if (((CudaNdarray_HOST_DIMS(B)[0] > 1) && (CudaNdarray_HOST_STRIDES(B)[0] != 1)) && ((CudaNdarray_HOST_DIMS(B)[1] > 1) && (CudaNdarray_HOST_STRIDES(B)[1] != 1)))
    { PyErr_SetString(PyExc_NotImplementedError, "non-unit stride in gemm arg"); return -1; }
    if (((CudaNdarray_HOST_DIMS(C)[0] > 1) && (CudaNdarray_HOST_STRIDES(C)[0] != 1)) && ((CudaNdarray_HOST_DIMS(C)[1] > 1) && (CudaNdarray_HOST_STRIDES(C)[1] != 1)))
    { PyErr_SetString(PyExc_NotImplementedError, "non-unit stride in gemm arg"); return -1; }

    // the unit integer is divided logically into three fields of 4 bits
    // the lowermost 4 bits encode the stride pattern of the output
    // the next higher 4 bits encode the B variable (or y)
    // the next higher 4 bits encode the C variable (or x)
    //
    // the stride pattern for each input is encoded as 0 for unit stride from col to col (Row major)
    //                                                 1 for unit stride from row to row (Col major)

    // a stride of 0 implies a dimension of 1 - so we can actually define
    // a stride of 0 as a 'unit' stride because gemm will never use it.
    int unit = 0;
    if (CudaNdarray_HOST_STRIDES(A)[1] == 1 || CudaNdarray_HOST_STRIDES(A)[1] == 0) {
        unit |= (0x0 << 8);
    } else if (CudaNdarray_HOST_STRIDES(A)[0] == 1 || CudaNdarray_HOST_STRIDES(A)[0] == 0) { 
        unit |= (0x1 << 8);
    } else {
        unit |= (0x2 << 8);
    }
    if (CudaNdarray_HOST_STRIDES(B)[1] == 1 || CudaNdarray_HOST_STRIDES(B)[1] == 0) {
        unit |= (0x0 << 4);
    } else if (CudaNdarray_HOST_STRIDES(B)[0] == 1 || CudaNdarray_HOST_STRIDES(B)[0] == 0) { 
        unit |= (0x1 << 4);
    } else {
        unit |= (0x2 << 4);
    }
    if (CudaNdarray_HOST_STRIDES(C)[1] == 1 || CudaNdarray_HOST_STRIDES(C)[1] == 0) {
        unit |= (0x0 << 0);
    } else if (CudaNdarray_HOST_STRIDES(C)[0] == 1 || CudaNdarray_HOST_STRIDES(C)[0] == 0) { 
        unit |= (0x1 << 0);
    } else {
        unit |= (0x2 << 0);
    }

    // I don't know if cudablas handles negative strides
    assert (CudaNdarray_HOST_STRIDES(A)[0] >= 0) ; // for now
    assert (CudaNdarray_HOST_STRIDES(A)[1] >= 0) ; // for now
    assert (CudaNdarray_HOST_STRIDES(B)[0] >= 0) ; // for now
    assert (CudaNdarray_HOST_STRIDES(B)[1] >= 0) ; // for now
    assert (CudaNdarray_HOST_STRIDES(C)[0] >= 0) ; // for now
    assert (CudaNdarray_HOST_STRIDES(C)[1] >= 0) ; // for now

    /* create appropriate strides for malformed matrices that are row or column
     * vectors
     */
    int sa_0 = (CudaNdarray_HOST_DIMS(A)[0] > 1) ? CudaNdarray_HOST_STRIDES(A)[0] : CudaNdarray_HOST_DIMS(A)[1];
    int sa_1 = (CudaNdarray_HOST_DIMS(A)[1] > 1) ? CudaNdarray_HOST_STRIDES(A)[1] : CudaNdarray_HOST_DIMS(A)[0];
    int sb_0 = (CudaNdarray_HOST_DIMS(B)[0] > 1) ? CudaNdarray_HOST_STRIDES(B)[0] : CudaNdarray_HOST_DIMS(B)[1];
    int sb_1 = (CudaNdarray_HOST_DIMS(B)[1] > 1) ? CudaNdarray_HOST_STRIDES(B)[1] : CudaNdarray_HOST_DIMS(B)[0];
    int sc_0 = (CudaNdarray_HOST_DIMS(C)[0] > 1) ? CudaNdarray_HOST_STRIDES(C)[0] : CudaNdarray_HOST_DIMS(C)[1];
    int sc_1 = (CudaNdarray_HOST_DIMS(C)[1] > 1) ? CudaNdarray_HOST_STRIDES(C)[1] : CudaNdarray_HOST_DIMS(C)[0];

    float* a = CudaNdarray_DEV_DATA(A);
    float* b = CudaNdarray_DEV_DATA(B);
    float* c = CudaNdarray_DEV_DATA(C);
    char N = 'N';
    char T = 'T';
    //std::cerr << (unit/256) MOD 16 << (unit / 16) MOD 16 << unit MOD 16<< '\\n';
    //TODO: recognize the negative stride and make a copy of the offending argument,
    //rather than aborting
#define CHK_STRIDE_SGEMM(T0, T1, D0, D1, D2, a, x, sx, y, sy, b, z, sz) \
    if ((sx > 0) && (sy > 0) && (sz > 0)) { \
        hipblasSgemm(T0, T1, D0, D1, D2, a, x, sx, y, sy, b, z, sz); \
    } else { \
        PyErr_SetString(PyExc_NotImplementedError, "negative stride to sGemm");\
        return -1; \
    } 

    switch(unit)
    {
        case 0x000: CHK_STRIDE_SGEMM(N, N, CudaNdarray_HOST_DIMS(C)[1], CudaNdarray_HOST_DIMS(C)[0], CudaNdarray_HOST_DIMS(A)[1], alpha, b, sb_0, a, sa_0, beta, c, sc_0); break;
        case 0x100: CHK_STRIDE_SGEMM(N, T, CudaNdarray_HOST_DIMS(C)[1], CudaNdarray_HOST_DIMS(C)[0], CudaNdarray_HOST_DIMS(A)[1], alpha, b, sb_0, a, sa_1, beta, c, sc_0); break;
        case 0x010: CHK_STRIDE_SGEMM(T, N, CudaNdarray_HOST_DIMS(C)[1], CudaNdarray_HOST_DIMS(C)[0], CudaNdarray_HOST_DIMS(A)[1], alpha, b, sb_1, a, sa_0, beta, c, sc_0); break;
        case 0x110: CHK_STRIDE_SGEMM(T, T, CudaNdarray_HOST_DIMS(C)[1], CudaNdarray_HOST_DIMS(C)[0], CudaNdarray_HOST_DIMS(A)[1], alpha, b, sb_1, a, sa_1, beta, c, sc_0); break;
        case 0x001: CHK_STRIDE_SGEMM(T, T, CudaNdarray_HOST_DIMS(C)[0], CudaNdarray_HOST_DIMS(C)[1], CudaNdarray_HOST_DIMS(A)[1], alpha, a, sa_0, b, sb_0, beta, c, sc_1); break;
        case 0x101: CHK_STRIDE_SGEMM(N, T, CudaNdarray_HOST_DIMS(C)[0], CudaNdarray_HOST_DIMS(C)[1], CudaNdarray_HOST_DIMS(A)[1], alpha, a, sa_1, b, sb_0, beta, c, sc_1); break;
        case 0x011: CHK_STRIDE_SGEMM(T, N, CudaNdarray_HOST_DIMS(C)[0], CudaNdarray_HOST_DIMS(C)[1], CudaNdarray_HOST_DIMS(A)[1], alpha, a, sa_0, b, sb_1, beta, c, sc_1); break;
        case 0x111: CHK_STRIDE_SGEMM(N, N, CudaNdarray_HOST_DIMS(C)[0], CudaNdarray_HOST_DIMS(C)[1], CudaNdarray_HOST_DIMS(A)[1], alpha, a, sa_1, b, sb_1, beta, c, sc_1); break;
        default: PyErr_Format(PyExc_ValueError, "some matrix has no unit stride (unit=%i)", unit);
                 return -1;
    };
    CNDA_THREAD_SYNC;
    if (HIPBLAS_STATUS_SUCCESS != cublasGetError())
    {
        PyErr_SetString(PyExc_RuntimeError, "cublassGemm failed");
        return -1;
    }
    return 0;
}

/**
 *
 * Precondition:
 *  a->dim[d] == (dims_a[d]==0) ? (1 << log2_dims_a[d]) : dims_a[d]
 *  z->dim[d] == (z_str[d]==0) ? 1 : dims_a[d];
 *
 *  TODO: templatize this function to support other reductions.
 *  All that needs to change is the initial value for sum, and the reduction operator.
 */

static __global__ void kernel_reduce_sum(const unsigned int size_z,
        const unsigned int nd,
        const int * dims_a,
        const int * log2_dims_a,
        const int * a_str,
        const float * a_data,
        const int * z_str,
        float * z_data)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    //structure data contains the strides and dimensions of both a and z
    // a_dim[0], a_dim[1], ... a_dim[nd-1],
    // a_log2dim[0], a_log2dim[1], ... a_log2dim[nd-1],
    // a_str[0], ... a_str[nd-1],
    // z_str[0], ... z_str[nd-1]
    extern __shared__ int structure_data[];
    for (unsigned int i = threadIdx.x; i < nd; i += blockDim.x)
    {
        structure_data[i+0*nd] = dims_a[i];
        structure_data[i+1*nd] = log2_dims_a[i];
        structure_data[i+2*nd] = a_str[i];
        structure_data[i+3*nd] = z_str[i];
    }
    dims_a = structure_data;
    log2_dims_a = structure_data + nd;
    a_str = structure_data + 2*nd;
    z_str = structure_data + 3*nd;

    __syncthreads(); //wait for all the shared structure to be loaded

    for (unsigned int i = idx; i < size_z; i += numThreads)
    {
        unsigned int ii = i;
        const float * a_data_i = a_data;
        float * z_data_i = z_data;
        unsigned int n_reduce_elements = 1;
        unsigned int n_reduce_dims = 0;
        unsigned int reduce_dim0 = nd-1;


        //In this loop, we locate the initial element of the slice that we'd like to reduce with this thread
        //  At the same time, we [re]calculate the size of that slice (n_reduce_elements)
        for (unsigned int d = 0; d < nd; ++d)
        {
            if (a_str[d] && (!z_str[d])) // this means 'd' is a dimension we are reducing over
            {
                n_reduce_elements *= dims_a[d];
                n_reduce_dims += 1;
                reduce_dim0 = (d < reduce_dim0) ? d : reduce_dim0;
            }
            else //'d' is not a dimension that we are reducing over
            {
                unsigned int pos_d;
                if (log2_dims_a[d]==-1) //TODO: when things are working, use this switch
                {
                    // this branch is not preferred, 
                    // because the manual said that integer mod and div operations are slow on gpu
                    pos_d = (ii % dims_a[d]);
                    ii = (ii / dims_a[d]);
                }
                else
                {
                    pos_d = (ii & ((1 << log2_dims_a[d])-1)); //take the lower log2_dims bits
                    ii = (ii >> log2_dims_a[d]);  //shift those lower log2_dims bits off of ii
                }
                a_data_i += pos_d * a_str[d];
                z_data_i += pos_d * z_str[d];
            }
        }
        // now we've got pointers a_data_i and z_data_i into element 0 of the slice over which we are reducing
        // do a similar loop
        
        float sum = 0.0f;
        switch(n_reduce_dims)
        {
            case 0:
                {
                    sum = a_data_i[0];
                } 
                break;
            case 1:
                {
                    const int stride = a_str[reduce_dim0];
                    const float * a_data_i_max = a_data_i + dims_a[reduce_dim0] * stride;
                    while (a_data_i != a_data_i_max)
                    {
                        sum += a_data_i[0];
                        a_data_i += stride;
                    }
                }
                break;
            case 2:
                {
                    int rd = reduce_dim0+1;
                    for (; rd < nd; ++rd)
                    {
                        if (a_str[rd] && (!z_str[rd])) // this means 'rd' is a dimension we are reducing over
                            break;
                    }
                    const int stride0 = a_str[reduce_dim0];
                    const int stride1 = a_str[rd];
                    for (int ii = 0; ii < dims_a[rd]; ++ii)
                    {
                        const float * a_data_ri = a_data_i + ii * stride1;
                        const float * a_data_ri_max = a_data_ri + dims_a[reduce_dim0] * stride0;
                        while (a_data_ri != a_data_ri_max)
                        {
                            sum += a_data_ri[0];
                            a_data_ri += stride0;
                        }
                    }
                }; 
                break;
            default:
                {
                    for (unsigned int reduce_i = 0; reduce_i < n_reduce_elements; ++reduce_i)
                    {
                        //TODO: optimize this loop to work more like theano's Elemwise.  It's serial code.
                        unsigned int reduce_ii = reduce_i;
                        const float * a_data_ri = a_data_i;

                        //This loop finds the element in the a slice to add.
                        for (unsigned int rd = reduce_dim0; rd < nd; ++rd)
                        {
                            unsigned int pos_d;
                            if (a_str[rd] && (!z_str[rd])) // this means 'd' is a dimension we are reducing over
                            {
                                if (log2_dims_a[rd]==-1)
                                {
                                    // this branch is not preferred, 
                                    // because the manual said that integer mod and div operations are slow on gpu
                                    pos_d = (reduce_ii % dims_a[rd]);
                                    reduce_ii = (reduce_ii / dims_a[rd]);
                                }
                                else
                                {
                                    pos_d = (reduce_ii & ((1 << log2_dims_a[rd])-1)); //take the lower log2_dims bits
                                    reduce_ii = (reduce_ii >> log2_dims_a[rd]);  //shift those lower log2_dims bits off of ii
                                }
                                a_data_ri += pos_d * a_str[rd];
                            }
                        }
                        sum += a_data_ri[0];
                    }
                }
        }
        z_data_i[0] = sum;
    }
}

static __global__ void kernel_reduce_sum_1011(
        const unsigned int d0,
        const unsigned int d1,
        const unsigned int d2,
        const unsigned int d3,
        const float *A, const int sA0, const int sA1, const int sA2, const int sA3,
        float * Z, const int sZ0)
{
    const int threadCount = blockDim.x * blockDim.y * blockDim.z;
    const int threadNum = threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    extern __shared__ float buf[];
    float mysum = 0.0f;

    if (warpSize != 32)
    {
        return;  //TODO: set error code
    }

    for (int i0 = threadIdx.z; i0 < d0; i0 += blockDim.z)
    {
        float Ai = A[i0 * sA0 + blockIdx.x * sA1 + threadIdx.y * sA2 + threadIdx.x * sA3];
        mysum += Ai;
    }
    buf[threadNum] = mysum;
    __syncthreads();

    // rest of function is handled by one warp
    if (threadNum < warpSize)
    {
        for (int i = threadNum + warpSize; i < threadCount; i += warpSize)
        {
            mysum += buf[i];
        }
        buf[threadNum] = mysum;
        if (threadNum < 16)
        {
            //reduce so that threadNum 0 has the sum of everything
            if(threadNum + 16 < threadCount) buf[threadNum] += buf[threadNum+16];
            if(threadNum + 8 < threadCount) buf[threadNum] += buf[threadNum+8];
            if(threadNum + 4 < threadCount) buf[threadNum] += buf[threadNum+4];
            if(threadNum + 2 < threadCount) buf[threadNum] += buf[threadNum+2];
            if(threadNum + 1 < threadCount) buf[threadNum] += buf[threadNum+1];
            if (threadNum == 0)
            {
                Z[blockIdx.x*sZ0] = buf[0];
            }
        }
    }
}
/**
 * Dimensions in which the self has size 1 and A has size > 1 are considered summing dimensions
 * Dimensions in which self has size > 1 and A has size > 1 are considered non-summing dimensions, and in this case their sizes must be equal.
 */
int 
CudaNdarray_reduce_sum(CudaNdarray * self, CudaNdarray * A)
{
    int verbose = 0;
    //check input rank
    if (self->nd != A->nd)
    {
        PyErr_Format(PyExc_TypeError, "Rank mismatch in CudaNdarray_sum: %i vs %i", self->nd, A->nd);
        return -1;
    }
    for (int i = 0; i < self->nd; ++i)
    {
        if ((CudaNdarray_HOST_DIMS(self)[i] > 1) && (CudaNdarray_HOST_DIMS(self)[i] != CudaNdarray_HOST_DIMS(A)[i]))
        {
            PyErr_Format(PyExc_TypeError, "Dimension mismatch in CudaNdarray_sum: self->dim[%i] == %i , A->dim[%i] = %i",
                    i, CudaNdarray_HOST_DIMS(self)[i], i, CudaNdarray_HOST_DIMS(A)[i]);
            return -1;
        }
    }

    int n_summations = (unsigned int)CudaNdarray_SIZE(self);
    if (verbose)
    {
        std::cerr << "reduce_sum n_summations " << n_summations  << '\n';
        std::cerr << "reduce_sum nd " << self->nd  << '\n';
        fprint_CudaNdarray(stderr, A);
        fprint_CudaNdarray(stderr, self);
    }
    if (0 && (A->nd == 4) //check to see if kernel_reduce_sum_1011 applies
            && (CudaNdarray_HOST_DIMS(self)[0] == 1)
            && (CudaNdarray_HOST_DIMS(self)[2] == 1)
            && (CudaNdarray_HOST_DIMS(self)[3] == 1)
       )
    {
        dim3 n_threads(CudaNdarray_HOST_DIMS(A)[3], CudaNdarray_HOST_DIMS(A)[2]);
        dim3 n_blocks(CudaNdarray_HOST_DIMS(A)[1]);
        while (n_threads.x * n_threads.y * n_threads.z < NUM_VECTOR_OP_THREADS_PER_BLOCK) ++n_threads.z;
        n_threads.z -= 1;
        if (n_threads.z > 64) n_threads.z = 64;
        if (n_threads.z)
        {
            if (verbose) printf("trying kernel_reduce_sum_1011\n");
            int n_shared = sizeof(float) * n_threads.x * n_threads.y * n_threads.z;
            kernel_reduce_sum_1011<<<n_blocks, n_threads, n_shared>>>(
                    CudaNdarray_HOST_DIMS(A)[0],
                    CudaNdarray_HOST_DIMS(A)[1],
                    CudaNdarray_HOST_DIMS(A)[2],
                    CudaNdarray_HOST_DIMS(A)[3],
                    CudaNdarray_DEV_DATA(A),
                    CudaNdarray_HOST_STRIDES(A)[0],
                    CudaNdarray_HOST_STRIDES(A)[1],
                    CudaNdarray_HOST_STRIDES(A)[2],
                    CudaNdarray_HOST_STRIDES(A)[3],
                    CudaNdarray_DEV_DATA(self),
                    CudaNdarray_HOST_STRIDES(self)[1]);
            CNDA_THREAD_SYNC;
            if (hipSuccess == hipGetLastError()) return 0;
            if (verbose) printf("failed, falling back to kernel_reduce_sum\n");
        }
    }

    int n_threads_per_block = std::min(n_summations,
            NUM_VECTOR_OP_THREADS_PER_BLOCK);
    int n_blocks = std::min(ceil_intdiv(n_summations,n_threads_per_block), 
            NUM_VECTOR_OP_BLOCKS);
    int n_structure_cache = self->nd * 4 * sizeof(int);

    if (verbose)
    {
        std::cerr << "n_blocks, n_threads_per_block " << n_blocks << ' ' << n_threads_per_block  << '\n';
    }
    assert (self->nd > 0);
    assert (self->nd == A->nd);
    kernel_reduce_sum<<<n_blocks, n_threads_per_block, n_structure_cache>>>(
            n_summations,
            self->nd,
            CudaNdarray_DEV_DIMS(A),
            CudaNdarray_DEV_LOG2DIMS(A),
            CudaNdarray_DEV_STRIDES(A),
            CudaNdarray_DEV_DATA(A),
            CudaNdarray_DEV_STRIDES(self),
            CudaNdarray_DEV_DATA(self));
    CNDA_THREAD_SYNC;
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) 
    {
        PyErr_Format(PyExc_RuntimeError, "Cuda error: %s: %s.\n", "kernel_reduce_sum", hipGetErrorString(err));
        return -1;
    }
    return 0;
}
int 
CudaNdarray_reduce_prod(CudaNdarray * self, const CudaNdarray * A)
{
    PyErr_SetString(PyExc_NotImplementedError, "");
    return -1;
}
int 
CudaNdarray_reduce_min(CudaNdarray * self, const CudaNdarray * A)
{
    PyErr_SetString(PyExc_NotImplementedError, "");
    return -1;
}
int 
CudaNdarray_reduce_max(CudaNdarray * self, const CudaNdarray * A)
{
    PyErr_SetString(PyExc_NotImplementedError, "");
    return -1;
}

/**
 *
 *  pattern is a permutation of [0, 1, ... self->nd-1] with the following twists:
 *  - an element 'd' of the permutation can be dropped if CudaNdarray_HOST_DIMS(self)[d] == 1
 *  - any number of '-1' elements can be in the pattern, and they will cause new ranks (with dim==1) to be inserted.
 *
 *  For example, if CudaNdarray_HOST_DIMS(self) == [4, 5, 1, 6], and pattern = [0,3,-1,-1, 1], then CudaNdarray_HOST_DIMS(self) would be modified to become:
 *     [4, 6, 1, 1, 5] (we dropped the original dim[2]==1, and inserted two singleton dimensions with the -1s.
 */
int 
CudaNdarray_dimshuffle(CudaNdarray * self, unsigned int len, const int * pattern)
{
    //TODO: pass a workspace pointer to avoid the internal malloc
    int * newdims = (int *)malloc(sizeof(int) * (len + len + self->nd)); //we tack on the taken buffer here for speed of not having to malloc twice.
    int * newstrides = newdims + len;
    int * dims_taken = newstrides + len;
    if (!newdims)
    {
        PyErr_SetString(PyExc_MemoryError, "CudaNdarray_dimshuffle: Failed to allocate temporary space");
        return -1;
    }
    for (int i = 0; i < self->nd; ++i)
    {
        dims_taken[i] = 0;
    }
    for (int i = 0; i < len; ++i)
    {
        if (pattern[i] < 0)
        {
            newdims[i] = 1;
            newstrides[i] = 0;
        }
        else
        {
            if ((dims_taken[pattern[i]]) or (pattern[i]>= self->nd))
            {
                PyErr_SetString(PyExc_ValueError, "invalid pattern for Cudandarray_dimshuffle");
                free(newdims);
                return -1;
            }
            newdims[i] = CudaNdarray_HOST_DIMS(self)[pattern[i]];
            newstrides[i] = CudaNdarray_HOST_STRIDES(self)[pattern[i]];
            dims_taken[pattern[i]] = 1;
        }
    }
    //swap this structure in for the one in self, and sync to the card
    if (CudaNdarray_set_nd(self, len))
    {
        free(newdims);
        return -1;
    }
    for (int i = 0; i < len; ++i)
    {
        CudaNdarray_set_dim(self, i, newdims[i]);
        CudaNdarray_set_stride(self, i, newstrides[i]);
    }
    if (cnda_copy_structure_to_device(self))
    {
        free(newdims);
        return -1;
    }
    free(newdims);
    return 0;
}

#include <conv_kernel.cu>
#include <conv_full_kernel.cu>

bool msgdisplayed_conv_patch__kern_width = false;
bool msgdisplayed_conv_patch_stack__kern_width = false;
bool msgdisplayed_conv_rows__kern_width = false;
bool msgdisplayed_conv_rows_stack__kern_width = false;
bool msgdisplayed_conv_rows_stack2__kern_width = false;
bool msgdisplayed_conv_patch_stack_reduce__kern_width = false;

bool msgdisplayed_conv_full_patch_stack__kern_width = false;

/*
 * version: -1, autodetect, >=0 a specific version to use.
 *          If it can't be executed, we revert to the reference implementation
 */
int 
CudaNdarray_conv_valid(const CudaNdarray *img, const CudaNdarray * kern,
		       CudaNdarray * out, int subsample_rows, int subsample_cols,
		       int version = -1, int verbose=0)
{
    int work_complete = 0;
    const int shared_avail = SHARED_SIZE-150;//144 is the biggest static shared size used with compiling this file.
    if (img->nd != 4)
    {
        PyErr_SetString(PyExc_ValueError, "required img of 4D");
        return -1;
    }
    if (kern->nd != 4)
    {
        PyErr_SetString(PyExc_ValueError, "required kern of 4D");
        return -1;
    }
    if (out->nd != 4)
    {
        PyErr_SetString(PyExc_ValueError, "required out of 4D");
        return -1;
    }
    if (subsample_rows==1 && subsample_cols==1)
    {
        //TODO: rethink these asserts in light of the difference between physical and logical dimensions
        assert (CudaNdarray_HOST_DIMS(out)[2] == CudaNdarray_HOST_DIMS(img)[2] - CudaNdarray_HOST_DIMS(kern)[2] + 1);
        assert (CudaNdarray_HOST_DIMS(out)[3] == CudaNdarray_HOST_DIMS(img)[3] - CudaNdarray_HOST_DIMS(kern)[3] + 1);
    }
    assert (CudaNdarray_HOST_DIMS(out)[0] == CudaNdarray_HOST_DIMS(img)[0]);
    assert (CudaNdarray_HOST_DIMS(out)[1] == CudaNdarray_HOST_DIMS(kern)[0]);
    assert (CudaNdarray_HOST_DIMS(img)[1] == CudaNdarray_HOST_DIMS(kern)[1]);

    // we now search through a few implementations until one applies to our arguments.
    
    //TODO: make separate version as if all fill this is slower. 
    //TODO: Make a switch with power of 2 max size as template
    //TODO: make a parameter the number of division
    //TODO: Should we make them in separate grid block instead?
 
    const int nstack=CudaNdarray_HOST_DIMS(kern)[1];
    const int nbatch=CudaNdarray_HOST_DIMS(img)[0];
    const int nkern=CudaNdarray_HOST_DIMS(kern)[0];
    const int img_wid=CudaNdarray_HOST_DIMS(img)[3];
    const int img_len=CudaNdarray_HOST_DIMS(img)[2];
    const int kern_wid=CudaNdarray_HOST_DIMS(kern)[3];
    const int kern_len=CudaNdarray_HOST_DIMS(kern)[2];
    const int out_wid=CudaNdarray_HOST_DIMS(out)[3];
    const int out_len=CudaNdarray_HOST_DIMS(out)[2];

    const int img_stride_col= CudaNdarray_HOST_STRIDES(img)[3];
    const int img_stride_row=CudaNdarray_HOST_STRIDES(img)[2];
    const int img_stride_stack= CudaNdarray_HOST_STRIDES(img)[1];
    const int img_stride_batch=CudaNdarray_HOST_STRIDES(img)[0];
    const int kern_stride_col= CudaNdarray_HOST_STRIDES(kern)[3];
    const int kern_stride_row=CudaNdarray_HOST_STRIDES(kern)[2];
    const int kern_stride_stack= CudaNdarray_HOST_STRIDES(kern)[1];
    const int kern_stride_nkern=CudaNdarray_HOST_STRIDES(kern)[0];

    const int img_size=img_len*img_wid;
    const int kern_size=kern_len*kern_wid;
    const int out_size=out_len*out_wid;
    const int img_size_byte = img_size*sizeof(float);
    const int kern_size_byte = kern_size*sizeof(float);
    const int out_size_byte = out_size*sizeof(float);

    bool subsample = subsample_rows!=1 || subsample_cols!=1;
    bool img_contiguous = CudaNdarray_is_c_contiguous(img);
    bool kern_contiguous = CudaNdarray_is_c_contiguous(kern);
    bool out_contiguous = CudaNdarray_is_c_contiguous(out);
    bool c_contiguous = img_contiguous &&  kern_contiguous && out_contiguous;

    bool img_contiguous_2d = (img_stride_col == 1) && (img_stride_row==img_wid);
    bool kern_contiguous_2d = (kern_stride_col == 1) && (kern_stride_row==kern_wid);

    //if the lower 2 dims are c_contiguous but flipped, unflipping the stride and not flipping the kernel in shared memroy
    //allow to use a version that use less registers(so is faster)
    //the unflipped version of variable haev the original value when we don't need to unflip it, but have the new value when we unflip it.
    bool kern_flipped=true;
    bool kern_contiguous_2d_unflipped = kern_contiguous_2d;
    float * kern_data_unflipped = kern->devdata;
    int kern_stride_col_unflipped=kern_stride_col;
    int kern_stride_row_unflipped=kern_stride_row;
    if(kern_stride_col_unflipped==-1 && kern_stride_row_unflipped==-kern_wid){
      //the last two dimensions are c_contiguous but flipped!
      kern_stride_col_unflipped=1;
      kern_stride_row_unflipped=kern_wid;
      kern_flipped=false;
      kern_contiguous_2d_unflipped = true;
      kern_data_unflipped=&(kern->devdata[(kern_wid-1)*kern_stride_col + (kern_len-1)*kern_stride_row]);
    }

    if (verbose>1)
    {
        printf("INFO: Running conv_valid version %d with inputs:\n",version);
        printf("INFO:   img  dim: %i %i %i %i  img  stride: %i %i %i %i\n", 
                CudaNdarray_HOST_DIMS(img)[0], CudaNdarray_HOST_DIMS(img)[1],CudaNdarray_HOST_DIMS(img)[2],CudaNdarray_HOST_DIMS(img)[3],
                CudaNdarray_HOST_STRIDES(img)[0], CudaNdarray_HOST_STRIDES(img)[1],CudaNdarray_HOST_STRIDES(img)[2],CudaNdarray_HOST_STRIDES(img)[3]);
        printf("INFO:   kern dim: %i %i %i %i  kern stride: %i %i %i %i\n",
                CudaNdarray_HOST_DIMS(kern)[0], CudaNdarray_HOST_DIMS(kern)[1],CudaNdarray_HOST_DIMS(kern)[2],CudaNdarray_HOST_DIMS(kern)[3],
                CudaNdarray_HOST_STRIDES(kern)[0], CudaNdarray_HOST_STRIDES(kern)[1],CudaNdarray_HOST_STRIDES(kern)[2],CudaNdarray_HOST_STRIDES(kern)[3]);
    }
    
    //if we remove the restriction img_size_byte+kern_size_byte>8*1024, we can enter in condition where we will lower the occupency due to shared memory and/or registers.
    if ((version == -1) && (out_size<64 || img_size_byte+kern_size_byte>8*1024) && out_size<=256){
      //condition for exec 
      if(!subsample &&
	out_contiguous &&
	out_size<512 &&//Maximum of 512 theads by block
	(img_size_byte+2*kern_wid*sizeof(float)+out_size_byte*2)<shared_avail && //their is only 16k of shared memory and if we can't have the output at least twice in shared mem, we won't have any reduce!
	!work_complete)
	version = 7; //conv_patch_stack_reduce, switch to version 8/13 automatically if needed.
    }

    if (!subsample && c_contiguous &&
	(version==0||version==2||version==-1) &&
	out_wid<512 &&//Maximum of 512 theads by block
	nstack == 1 &&// don't implement the stack in the kernel.
	img_size_byte+kern_size_byte<shared_avail && //their is only 16k of shared memory
	!work_complete) //conv_patch
    {
        int nb_split=1;//The number of split (i.e. the number of output pixel each thread compute.)
	if(version==2 && out_len>1)nb_split++;//to force the use of split=true when testing.
	//we pass by ceil_intdiv in case the out_len is not a multiple of nb_split, we want nb_split the number of iteration.
	while (ceil_intdiv(out_len,nb_split)*out_wid>512) nb_split++;
        dim3 threads(out_wid, ceil_intdiv(out_len,nb_split));

        dim3 grid(nbatch, nkern);
        int shared_size=(img_size + kern_size)*sizeof(float);
	void (*f)(float*, float*, float*,
		  int, int, int, int,
		  int, int);

#define CONV_PATCH_SPECIAL(kern_wid) \
            if(threads.y==out_len) f=conv_patch<true,kern_wid,false>;\
            else f=conv_patch<true,kern_wid,true>;

	 switch(kern_wid){
#ifdef UNROLL_LOOP
	 case 1: CONV_PATCH_SPECIAL(1); break;//test_conv.py:test_valid
	 case 2: CONV_PATCH_SPECIAL(2); break;//test_conv.py:test_valid
	 case 3: CONV_PATCH_SPECIAL(3); break;//test_conv.py:test_valid
	 case 4: CONV_PATCH_SPECIAL(4); break;
	 case 5: CONV_PATCH_SPECIAL(5); break;
	 case 6: CONV_PATCH_SPECIAL(6); break;
	 case 7: CONV_PATCH_SPECIAL(7); break;
	 case 10: CONV_PATCH_SPECIAL(10); break;
#endif
	 default:
	   if(!msgdisplayed_conv_patch__kern_width) {
	     printf("OPTIMISATION WARNING: conv_patch template default add kern_wid=%d in %s at line %i to have an optimized version for your kern_wid\n", kern_wid, __FILE__, __LINE__);
	     msgdisplayed_conv_patch__kern_width=true;
	   }
	   CONV_PATCH_SPECIAL(0);
	 }
	 f<<< grid, threads, shared_size>>>
	     (img->devdata, kern->devdata, out->devdata,
	      img_len, img_wid, kern_len, kern_wid, nkern, nstack);
        CNDA_THREAD_SYNC;
        hipError_t sts = hipGetLastError();
        if (hipSuccess == sts) 
        {
            if (verbose) printf("INFO: used 'conv_patch' version %s nb_split=%d\n",threads.y==out_len?"no split": "split",nb_split);
            work_complete = true;
        }
        else
        {
            if (verbose) printf("threads.x=%i, threads.y=%i, grid.x=%i, grid.y=%i, shared_size=%i, nb_threads=%i, nb_split=%i\n", threads.x, threads.y, grid.x, grid.y, shared_size, threads.x * threads.y, nb_split);
            if (verbose) printf("INFO: impl 'conv_patch' failed (%s), trying next implementation\n",
                                hipGetErrorString(sts));
        }                         
    }
    if (!subsample &&
	out_contiguous &&
	(version==1||version==3||version==11||version==12||version==-1) &&
	(version!=1 || out_size<512) &&//Maximum of 512 theads by block
	out_wid<512 &&//Maximum of 512 theads by block
	img_size_byte+kern_wid*sizeof(float)<shared_avail && //their is only 16k of shared memory
	!work_complete) //conv_patch_stack
    {
      //version 1 is without split and preload the full kernel
      //version 3 is with split and preload the full kernel
      //version 11 is without split and load only 1 kernel row at a time.
      //version 12 is with split and load only 1 kernel row at a time.
        int nb_split=1;//The number of split (i.e. the number of output pixel each thread compute.)
	if((version==3||version==12) && out_len>1)nb_split++;//to force the use of split=true when testing.
	//we pass by ceil_intdiv in case the out_len is not a multiple of nb_split, we want nb_split the number of iteration.
	while (ceil_intdiv(out_len,nb_split)*out_wid>512) nb_split++;
        dim3 threads(out_wid, ceil_intdiv(out_len,nb_split));

	bool preload_full_kernel = (img_size_byte + kern_size_byte) <shared_avail;
	if(version==11 || version==12) preload_full_kernel=false;
        dim3 grid(nbatch,nkern);
        int shared_size=(img_size + (preload_full_kernel?kern_size:kern_wid))*sizeof(float);

	void (*f)(float*, float*, float*,
		  int, int, int, int,
		  int, int, int, int,
		  int, int, int, int,
		  int, int);

#define CONV_PATCH_STACK_SPECIAL(kern_wid) \
            if(preload_full_kernel && nb_split==1 && img_contiguous_2d && kern_contiguous_2d) f=conv_patch_stack<true,false,kern_wid,true,true,false,true>;\
            if(preload_full_kernel && nb_split==1 && img_contiguous_2d && !kern_contiguous_2d) f=conv_patch_stack<true,false,kern_wid,true,false,false,true>;\
            if(preload_full_kernel && nb_split==1 && !img_contiguous_2d && kern_contiguous_2d) f=conv_patch_stack<true,false,kern_wid,false,true,false,true>;\
            if(preload_full_kernel && nb_split==1 && !img_contiguous_2d && !kern_contiguous_2d) f=conv_patch_stack<true,false,kern_wid,false,false,false,true>;\
            if(preload_full_kernel && img_contiguous_2d && kern_contiguous_2d) f=conv_patch_stack<true,false,kern_wid,true,true,true,true>;\
            if(preload_full_kernel && img_contiguous_2d && !kern_contiguous_2d) f=conv_patch_stack<true,false,kern_wid,true,false,true,true>;\
            if(preload_full_kernel && !img_contiguous_2d && kern_contiguous_2d) f=conv_patch_stack<true,false,kern_wid,false,true,true,true>;\
            if(preload_full_kernel && !img_contiguous_2d && !kern_contiguous_2d) f=conv_patch_stack<true,false,kern_wid,false,false,true,true>;\
            if(nb_split==1 && img_contiguous_2d && kern_contiguous_2d) f=conv_patch_stack<true,false,kern_wid,true,true,false,false>;\
            if(nb_split==1 && img_contiguous_2d && !kern_contiguous_2d) f=conv_patch_stack<true,false,kern_wid,true,false,false,false>;\
            if(nb_split==1 && !img_contiguous_2d && kern_contiguous_2d) f=conv_patch_stack<true,false,kern_wid,false,true,false,false>;\
            if(nb_split==1 && !img_contiguous_2d && !kern_contiguous_2d) f=conv_patch_stack<true,false,kern_wid,false,false,false,false>;\
            if(img_contiguous_2d && kern_contiguous_2d) f=conv_patch_stack<true,false,kern_wid,true,true,true,false>;\
            if(img_contiguous_2d && !kern_contiguous_2d) f=conv_patch_stack<true,false,kern_wid,true,false,true,false>;\
            if(!img_contiguous_2d && kern_contiguous_2d) f=conv_patch_stack<true,false,kern_wid,false,true,true,false>;\
            if(!img_contiguous_2d && !kern_contiguous_2d) f=conv_patch_stack<true,false,kern_wid,false,false,true,false>;
	switch(kern_wid){
#ifdef UNROLL_LOOP
         case 1: CONV_PATCH_STACK_SPECIAL(1); break;
         case 2: CONV_PATCH_STACK_SPECIAL(2); break;
         case 3: CONV_PATCH_STACK_SPECIAL(3); break;
         case 4: CONV_PATCH_STACK_SPECIAL(4); break;
         case 5: CONV_PATCH_STACK_SPECIAL(5); break;
         case 6: CONV_PATCH_STACK_SPECIAL(6); break;
         case 7: CONV_PATCH_STACK_SPECIAL(7); break;
         case 8: CONV_PATCH_STACK_SPECIAL(8); break;
         case 9: CONV_PATCH_STACK_SPECIAL(9); break;
         case 10: CONV_PATCH_STACK_SPECIAL(10); break;
                  //////// Special cases
         case 12: CONV_PATCH_STACK_SPECIAL(12); break;//on cifar10
         case 21: CONV_PATCH_STACK_SPECIAL(21); break;//on cifar10
         case 23: CONV_PATCH_STACK_SPECIAL(23); break;//test_nnet.py:test_lenet_64
         case 24: CONV_PATCH_STACK_SPECIAL(24); break;//on cifar10
         case 25: CONV_PATCH_STACK_SPECIAL(25); break;//on cifar10
         case 28: CONV_PATCH_STACK_SPECIAL(28); break;
         case 32: CONV_PATCH_STACK_SPECIAL(32); break;// Alex speed example
         case 45: CONV_PATCH_STACK_SPECIAL(45); break;//used by test_nnet.py:test_lenet_108
#endif
                  //////// default case
        default:
            if(!msgdisplayed_conv_patch_stack__kern_width) {
                printf("OPTIMISATION HINT: conv_patch_stack template default add kern_wid=%d in %s at line %i to have an optimized version for your kern_wid\n", kern_wid, __FILE__, __LINE__);
                msgdisplayed_conv_patch_stack__kern_width = true;
            }
            CONV_PATCH_STACK_SPECIAL(0);
	}
	f<<< grid, threads, shared_size>>>
	     (img->devdata, kern->devdata, out->devdata,
	      img_len, img_wid, kern_len, kern_wid, nkern, nstack,
	      img_stride_col, img_stride_row, img_stride_stack,
	      img_stride_batch, kern_stride_col, kern_stride_row,
	      kern_stride_stack, kern_stride_nkern);

        CNDA_THREAD_SYNC;
        hipError_t sts = hipGetLastError();
        if (hipSuccess == sts) 
        {
            if (verbose>1) printf("threads.x=%i, threads.y=%i, grid.x=%i, grid.y=%i,shared_size=%i, nb_threads=%i, nb_split=%i preload_full_kernel=%i\n",
				threads.x, threads.y, grid.x, grid.y, shared_size, threads.x * threads.y, nb_split, preload_full_kernel);
            if (verbose) printf("INFO: used 'conv_patch_stack' version with nb_split=%i and preload_full_kernel=%i\n",
				nb_split,preload_full_kernel);
            work_complete = true;
        }
        else
        {
            if (verbose) printf("threads.x=%i, threads.y=%i, grid.x=%i, grid.y=%i,shared_size=%i, nb_threads=%i, nb_split=%i preload_full_kernel=%i\n",
				threads.x, threads.y, grid.x, grid.y, shared_size, threads.x * threads.y, nb_split, preload_full_kernel);
            if (verbose) printf("INFO: impl 'conv_patch_stack' failed (%s), trying next implementation\n",
                                hipGetErrorString(sts));
        }                         
    }

    if (!subsample && out_contiguous &&
	(version==4||version==-1) &&
	out_wid<512 &&//Maximum of 512 threads by block
	nstack == 1 &&// don't implement the stack in the kernel.
	kern_len*img_wid*sizeof(float)+kern_size_byte<shared_avail &&//their is only 16k of shared memory
	!work_complete) //conv_rows

    {
        dim3 threads(out_wid);
        dim3 grid(out_len, nbatch*nkern);
        int shared_size=(kern_len*img_wid + kern_size)*sizeof(float);
	void (*f)(float*, float*, float*,
		  int, int, int, int,
		  int, int, int, int,
		  int, int, int, int,
		  int, int);

#define CONV_ROWS_SPECIAL(kern_wid) \
	if(!img_contiguous_2d || !kern_contiguous_2d) f = conv_rows<kern_wid, false>;\
	else f = conv_rows<kern_wid, true>;\

	switch(kern_wid){
#ifdef UNROLL_LOOP
	case 1: CONV_ROWS_SPECIAL(1); break;//test_conv.py:test_valid
	case 2: CONV_ROWS_SPECIAL(2); break;//test_conv.py:test_valid
	case 3: CONV_ROWS_SPECIAL(3); break;//test_conv.py:test_valid
	case 4: CONV_ROWS_SPECIAL(4); break;//test_conv.py:test_valid
	case 5: CONV_ROWS_SPECIAL(5); break;//test_conv.py:test_valid
	  //	case 6: CONV_ROWS_SPECIAL(6); break;
	case 7: CONV_ROWS_SPECIAL(7); break;//used by test_nnet.py:test_lenet_108
	  //	case 8: CONV_ROWS_SPECIAL(8); break;
	case 9: CONV_ROWS_SPECIAL(9); break;//used by test_nnet.py:test_lenet_256
	case 10: CONV_ROWS_SPECIAL(10); break;//test_conv.py:test_valid
	  //////// Special cases
	case 28: CONV_ROWS_SPECIAL(28); break;
#endif
	  //////// default case
	default:
	  if(!msgdisplayed_conv_rows__kern_width){
	    printf("OPTIMISATION HINT: conv_rows template default add kern_wid=%d in %s at line %i to have an optimized version for your kern_wid\n", kern_wid, __FILE__, __LINE__);
	    msgdisplayed_conv_rows__kern_width = true;
	  }
	  CONV_ROWS_SPECIAL(0);
	}

	f<<< grid, threads, shared_size >>>
	  (img->devdata, kern->devdata, out->devdata,
	   img_len, img_wid, kern_len, kern_wid, nkern, nstack,
	   img_stride_col, img_stride_row,
	   img_stride_stack,img_stride_batch,
	   kern_stride_col, kern_stride_row,
	   kern_stride_stack, kern_stride_nkern);

        CNDA_THREAD_SYNC;
        hipError_t sts = hipGetLastError();
        if (hipSuccess == sts) 
        {
            work_complete = true;
            if (verbose) printf("INFO: used 'conv_rows' version\n");
        }
        else
        {
            if (verbose) printf("threads.x=%i, threads.y=%i, grid.x=%i, grid.y=%i, shared_size=%i, nb_threads=%i\n", threads.x, threads.y, grid.x, grid.y, shared_size, threads.x * threads.y);
            if (verbose) printf("INFO: impl 'conv_rows' failed (%s), trying next implementation\n",
                    hipGetErrorString(sts));
        }                         
    }
    if (!subsample && out_contiguous &&
	(version==5||version==-1) &&
	out_wid<512 &&//Maximum of 512 theads by block
	img_wid*kern_len*sizeof(float)+kern_size_byte<shared_avail && //their is only 16k of shared memory
	!work_complete) //conv_rows_stack

    {
	int nb_row=1;
	int max_threads=512;
	//TODO:if not c_contiguous, lower max_thread as we use 22 registers by thread and we won't execute 2 block in one MP.
	for(int i=2;i<=out_len;i++){
	  if((i)*out_wid<max_threads && ((kern_len+i)*img_wid + kern_size)*sizeof(float)<shared_avail)
	    nb_row=i;
	}

        dim3 threads(out_wid,nb_row);
        dim3 grid(ceil_intdiv(out_len,nb_row), nbatch*nkern);
	  
        int shared_size=((kern_len+nb_row-1)*img_wid + kern_size)*sizeof(float);

	void (*f)(float*, float*, float*,
		  int, int, int, int,
		  int, int, int, int,
		  int, int, int, int,
		  int, int);

#define CONV_ROWS_STACK_SPECIAL(kern_wid) \
	if(!img_contiguous_2d || !kern_contiguous_2d) f = conv_rows_stack<kern_wid, false>;\
	else f = conv_rows_stack<kern_wid, true>;\

	switch(kern_wid){
#ifdef UNROLL_LOOP
	case 1: CONV_ROWS_STACK_SPECIAL(1); break;//test_conv.py:test_valid
	case 2: CONV_ROWS_STACK_SPECIAL(2); break;
	case 3: CONV_ROWS_STACK_SPECIAL(3); break;//test_conv.py:test_valid
	case 4: CONV_ROWS_STACK_SPECIAL(4); break;//test_conv.py:test_valid
	case 5: CONV_ROWS_STACK_SPECIAL(5); break;//test_conv.py:test_valid
	case 6: CONV_ROWS_STACK_SPECIAL(6); break;//test_conv.py:test_valid
	case 7: CONV_ROWS_STACK_SPECIAL(7); break;//test_nnet.py:test_lenet_108
	case 8: CONV_ROWS_STACK_SPECIAL(8); break;//test_conv.py:test_valid
	case 9: CONV_ROWS_STACK_SPECIAL(9); break;//test_nnet.py:test_lenet_256
	case 10: CONV_ROWS_STACK_SPECIAL(10); break;//test_conv.py:test_valid
	  //////// Special cases
	case 23: CONV_ROWS_STACK_SPECIAL(23); break;//test_conv.py:test_valid
	case 24: CONV_ROWS_STACK_SPECIAL(24); break;//test_conv.py:test_valid
	case 28: CONV_ROWS_STACK_SPECIAL(28); break;//test_conv.py:test_valid
	case 45: CONV_ROWS_STACK_SPECIAL(45); break;//test_nnet.py:test_lenet_64
	case 102: CONV_ROWS_STACK_SPECIAL(102); break;//test_nnet.py:test_lenet_108
#endif
	  //////// default case
	default:
	  if(!msgdisplayed_conv_rows_stack__kern_width){
	    printf("OPTIMISATION HINT: conv_rows_stack template default add kern_wid=%d in %s at line %i to have an optimized version for your kern_wid\n", kern_wid, __FILE__, __LINE__);
	    msgdisplayed_conv_rows_stack__kern_width = true;
	  }
	  CONV_ROWS_STACK_SPECIAL(0);
	}

	f<<< grid, threads, shared_size >>>
	  (img->devdata,
	   kern->devdata,
	   out->devdata,
	   img_len, img_wid, kern_len, kern_wid, nkern, nstack,
	   img_stride_col, img_stride_row,
	   img_stride_stack,img_stride_batch,
	   kern_stride_col, kern_stride_row,
	   kern_stride_stack, kern_stride_nkern);

        CNDA_THREAD_SYNC;
        hipError_t sts = hipGetLastError();
        if (hipSuccess == sts) 
        {
            work_complete = true;
	    if (verbose>1) printf("threads.x=%i, threads.y=%i, grid.x=%i, grid.y=%i, shared_size=%i, nb_threads=%i\n", threads.x, threads.y, grid.x, grid.y, shared_size, threads.x * threads.y);
            if (verbose) printf("INFO: used 'conv_rows_stack' version\n");
        }
        else
        {
            if (verbose) printf("threads.x=%i, threads.y=%i, grid.x=%i, grid.y=%i, shared_size=%i, nb_threads=%i\n", threads.x, threads.y, grid.x, grid.y, shared_size, threads.x * threads.y);
            if (verbose) printf("INFO: impl 'conv_rows_stack' failed (%s), trying next implementation\n",
		     hipGetErrorString(sts));
        }                         
    }

    if (!subsample && out_contiguous &&
	(version==9||version==10||version==-1) &&
	out_wid<512 &&//Maximum of 512 threads by block
	(img_wid+kern_wid)*sizeof(float)<shared_avail && //their is only 16k of shared memory
	(version != 9 || (img_wid+kern_len*kern_wid)*sizeof(float)<shared_avail) && //version 9 use more memory
	!work_complete) //conv_rows_stack2

    {
	int nb_row=1;
	int max_threads=512;
	int version_back = version;
	//TODO:if not c_contiguous, lower max_thread as we use 22 registers by thread and we won't execute 2 block in one MP.
	if(version==-1 && (img_wid+kern_len*kern_wid)*sizeof(float)<shared_avail)
	  version = 9;
	else if(version==-1)version = 10;

	int k_size = kern_size;
	if(version==10)
	  k_size=kern_wid;

	for(int i=2;i<=out_len;i++){
	  if(i*out_wid<max_threads && (i*img_wid + k_size)*sizeof(float)<shared_avail)
	    nb_row=i;
	}

	//to test the case when we don't have a thread by output pixel.
	if((version_back!=-1)&& nb_row>1) nb_row--;

        dim3 threads(out_wid,nb_row);
        dim3 grid(ceil_intdiv(out_len,nb_row), nbatch*nkern);
	  
        int shared_size=(threads.y*img_wid + k_size)*sizeof(float);

	void (*f)(float*, float*, float*,
		  int, int, int, int,
		  int, int, int, int,
		  int, int, int, int,
		  int, int);

#define CONV_ROWS_STACK2_SPECIAL(kern_wid) \
	if((!img_contiguous_2d || !kern_contiguous_2d)&&version==9) f = conv_rows_stack2<kern_wid, false,true>;\
	else if(version==9) f = conv_rows_stack2<kern_wid, true,true>;\
	else if(!img_contiguous_2d || !kern_contiguous_2d) f = conv_rows_stack2<kern_wid, false, false>;\
	else f = conv_rows_stack2<kern_wid, true, false>;\

	switch(kern_wid){
#ifdef UNROLL_LOOP
	case 1: CONV_ROWS_STACK2_SPECIAL(1); break;//test_conv.py:test_valid
	case 2: CONV_ROWS_STACK2_SPECIAL(2); break;
	case 3: CONV_ROWS_STACK2_SPECIAL(3); break;//test_conv.py:test_valid
	case 4: CONV_ROWS_STACK2_SPECIAL(4); break;//test_conv.py:test_valid
	case 5: CONV_ROWS_STACK2_SPECIAL(5); break;//test_conv.py:test_valid
	case 6: CONV_ROWS_STACK2_SPECIAL(6); break;//test_conv.py:test_valid
	case 7: CONV_ROWS_STACK2_SPECIAL(7); break;//test_nnet.py:test_lenet_108
	case 8: CONV_ROWS_STACK2_SPECIAL(8); break;//test_conv.py:test_valid
	  //	case 9: CONV_ROWS_STACK2_SPECIAL(9); break;
	case 10: CONV_ROWS_STACK2_SPECIAL(10); break;//test_conv.py:test_valid
	  //////// Special cases
	case 23: CONV_ROWS_STACK2_SPECIAL(23); break;//test_conv.py:test_valid
	case 24: CONV_ROWS_STACK2_SPECIAL(24); break;//test_conv.py:test_valid
	case 28: CONV_ROWS_STACK2_SPECIAL(28); break;//test_conv.py:test_valid
	case 45: CONV_ROWS_STACK2_SPECIAL(45); break;//test_nnet.py:test_lenet_108
	case 58: CONV_ROWS_STACK2_SPECIAL(58); break;//test_nnet.py:test_lenet_108
	case 70: CONV_ROWS_STACK2_SPECIAL(70); break;//mobahi_2009.py
	case 102: CONV_ROWS_STACK2_SPECIAL(102); break;//test_nnet.py:test_lenet_108
	case 116: CONV_ROWS_STACK2_SPECIAL(116); break;//test_nnet.py:test_lenet_256
	case 248: CONV_ROWS_STACK2_SPECIAL(248); break;//test_nnet.py:test_lenet_256
#endif
	  //////// default case
	default:
	  if(!msgdisplayed_conv_rows_stack2__kern_width){
	    printf("OPTIMISATION HINT: conv_rows_stack{2,3} template default add"
		   " kern_wid=%d in %s at line %i to have an optimized version"
		   " for your kern_wid\n", kern_wid, __FILE__, __LINE__);
	    msgdisplayed_conv_rows_stack2__kern_width = true;
	  }
	  CONV_ROWS_STACK2_SPECIAL(0);
	}

	f<<< grid, threads, shared_size >>>
	  (img->devdata,
	   kern->devdata,
	   out->devdata,
	   img_len, img_wid, kern_len, kern_wid, nkern, nstack,
	   img_stride_col, img_stride_row,
	   img_stride_stack,img_stride_batch,
	   kern_stride_col, kern_stride_row,
	   kern_stride_stack, kern_stride_nkern);

        CNDA_THREAD_SYNC;
        hipError_t sts = hipGetLastError();
        if (hipSuccess == sts) 
        {
            work_complete = true;
	    if (verbose>1) printf("threads.x=%i, threads.y=%i, grid.x=%i, grid.y=%i, shared_size=%i, nb_threads=%i\n",
				  threads.x, threads.y, grid.x, grid.y, shared_size, threads.x * threads.y);
            if (verbose) printf("INFO: used 'conv_rows_stack2' version %s with %d row(s).\n",(version==9?"'load full kernel'":"'load 1 kern row at a time'"),nb_row);
        }
        else
        {
            if (verbose) printf("threads.x=%i, threads.y=%i, grid.x=%i, grid.y=%i, shared_size=%i, nb_threads=%i version=%d\n",
				threads.x, threads.y, grid.x, grid.y, shared_size, threads.x * threads.y,(version==9?2:3));
            if (verbose) printf("INFO: impl 'conv_rows_stack2' failed (%s), trying next implementation\n",
		     hipGetErrorString(sts));
        }                         
    }

    //version 8 is the same but we force the split. The split is need in case we have too much threads. This happen frequently if the kernel length is big. Big kernel is frequent in the gradient.
    //version 8 need a minimum of kernel length as we force the split.
    //version 8 is needed to test more easily this kernel template parameter.
    //version 13 load only 1 kernel row at a time.
    if (!subsample &&
	out_contiguous &&
	out_size<512 &&//Maximum of 512 theads by block
	(version==7||version==8||version==13||version==-1) &&
	(version!=8||kern_len>1) && //version 8 need a minimal kernel length as big as the split.
	(version!=13||kern_len>1) && //version 13 need a minimal kernel length as big as the split.
	(img_size_byte+2*kern_wid*sizeof(float)+out_size_byte*2)<shared_avail && //their is only 16k of shared memory and if we can't have the output at least twice in shared mem, we won't have any reduce!
	!work_complete) //conv_patch_stack_reduce
    {
	int nb_split=1;
	int full_kern=true;

	if(version==8||version==13) nb_split++;//force the split.
	if(version==13)full_kern=false;
	while(ceil_intdiv(kern_len,nb_split)>64)nb_split++;//device 1.3 have a max of 64 thread in z
	while(out_size*ceil_intdiv(kern_len,nb_split)>512)nb_split++;
	int shared_size=(img_size + kern_size + out_size*kern_len)*sizeof(float);
	if(shared_size>=shared_avail){
	  //if we can't fit the kernel in shared memory, we can split it more.
	  full_kern=false;	  
	  assert((img_size+kern_wid*2+out_size*2)*sizeof(float)<=shared_avail);
	  shared_size=(img_size + kern_wid*ceil_intdiv(kern_len,nb_split) + out_size*ceil_intdiv(kern_len,nb_split))*sizeof(float);
	  while(shared_size>=shared_avail || ceil_intdiv(kern_len,nb_split)>64){
	    nb_split++;
	    shared_size=(img_size + kern_wid*ceil_intdiv(kern_len,nb_split) + out_size*ceil_intdiv(kern_len,nb_split))*sizeof(float);
	  }
	}

	int thread_z=ceil_intdiv(kern_len,nb_split);
        assert(thread_z>0);//should not happen, but in case...
	assert(shared_size<=shared_avail);
	if(!full_kern)
	  assert(thread_z!=kern_len);

        dim3 threads(out_wid, out_len, thread_z);
        dim3 grid(nbatch,nkern);
	
	void (*f)(float*, float*, float*,
		  int, int, int, int,
		  int, int, int, int,
		  int, int,
		  int, int,
		  int, int);

	const bool split=thread_z!=kern_len;
	const bool ccontig=img_contiguous_2d && kern_contiguous_2d_unflipped;

	//printf("kern_flipped=%d, ccontig=%d, split=%d, full_kern=%d\n",kern_flipped,ccontig,split,full_kern);
	//We will always be split when we don't load the full kernel
#define CONV_PATCH_STACK_REDUCE_SPECIAL(kern_wid) \
            if     (kern_flipped  && ccontig  && !split && full_kern) f=conv_patch_stack_reduce<true,kern_wid,true, false, true>;\
            else if(kern_flipped  && !ccontig && !split && full_kern) f=conv_patch_stack_reduce<true,kern_wid,false, false, true>;\
            else if(kern_flipped  && ccontig  && split && full_kern) f=conv_patch_stack_reduce<true,kern_wid,true, true, true>;\
            else if(kern_flipped  && !ccontig && split && full_kern) f=conv_patch_stack_reduce<true,kern_wid,false, true, true>;\
            else if(!kern_flipped && ccontig  && !split && full_kern) f=conv_patch_stack_reduce<false,kern_wid,true, false, true>;\
            else if(!kern_flipped && !ccontig && !split && full_kern) f=conv_patch_stack_reduce<false,kern_wid,false, false, true>;\
            else if(!kern_flipped && ccontig  && split && full_kern) f=conv_patch_stack_reduce<false,kern_wid,true, true, true>;\
            else if(!kern_flipped && !ccontig  && split && full_kern) f=conv_patch_stack_reduce<false,kern_wid,false, true, true>;\
	    /*else if(kern_flipped  && ccontig  && !split && !full_kern) f=conv_patch_stack_reduce<true,kern_wid,true, false, false>;*/\
	    /*else if(kern_flipped  && !ccontig && !split && !full_kern) f=conv_patch_stack_reduce<true,kern_wid,false, false, false>;*/\
            else if(kern_flipped  && ccontig  && split && !full_kern) f=conv_patch_stack_reduce<true,kern_wid,true, true, false>;\
            else if(kern_flipped  && !ccontig && split && !full_kern) f=conv_patch_stack_reduce<true,kern_wid,false, true, false>;\
            /*else if(!kern_flipped && ccontig  && !split && !full_kern) f=conv_patch_stack_reduce<false,kern_wid,true, false, false>;*/\
            /*else if(!kern_flipped && !ccontig && !split && !full_kern) f=conv_patch_stack_reduce<false,kern_wid,false, false, false>;*/\
            else if(!kern_flipped && ccontig  && split && !full_kern) f=conv_patch_stack_reduce<false,kern_wid,true, true, false>;\
            else if(!kern_flipped && !ccontig  && split && !full_kern) f=conv_patch_stack_reduce<false,kern_wid,false, true, false>;
	switch(kern_wid){
#ifdef UNROLL_LOOP
         case 1: CONV_PATCH_STACK_REDUCE_SPECIAL(1); break;
         case 2: CONV_PATCH_STACK_REDUCE_SPECIAL(2); break;
         case 3: CONV_PATCH_STACK_REDUCE_SPECIAL(3); break;
         case 4: CONV_PATCH_STACK_REDUCE_SPECIAL(4); break;
         case 5: CONV_PATCH_STACK_REDUCE_SPECIAL(5); break;
         case 6: CONV_PATCH_STACK_REDUCE_SPECIAL(6); break;
         case 7: CONV_PATCH_STACK_REDUCE_SPECIAL(7); break;
         case 8: CONV_PATCH_STACK_REDUCE_SPECIAL(8); break;
         case 9: CONV_PATCH_STACK_REDUCE_SPECIAL(9); break;
         case 10: CONV_PATCH_STACK_REDUCE_SPECIAL(10); break;
                  //////// Special cases
         case 20: CONV_PATCH_STACK_REDUCE_SPECIAL(20); break;
         case 23: CONV_PATCH_STACK_REDUCE_SPECIAL(23); break;//test_nnet.py:test_lenet64
         case 24: CONV_PATCH_STACK_REDUCE_SPECIAL(24); break;
         case 28: CONV_PATCH_STACK_REDUCE_SPECIAL(28); break;
         case 32: CONV_PATCH_STACK_REDUCE_SPECIAL(32); break;//Alex speed demonstration
#endif
                  //////// default case
        default:
            if(!msgdisplayed_conv_patch_stack_reduce__kern_width) {
                printf("OPTIMISATION HINT: conv_patch_stack_reduce template default add kern_wid=%d in %s at line %i to have an optimized version for your kern_wid\n", kern_wid, __FILE__, __LINE__);
                msgdisplayed_conv_patch_stack_reduce__kern_width = true;
            }
            CONV_PATCH_STACK_REDUCE_SPECIAL(0);
	}

	if (verbose) printf("INFO: using 'conv_patch_stack_reduce' version nb_split=%d, preload_full_kern=%d\n",
			    nb_split,full_kern);
	if (verbose>1) printf("threads.x=%i, threads.y=%i, threads.z=%i, grid.x=%i, grid.y=%i,shared_size=%i, nb_threads=%i\n",
			      threads.x, threads.y, threads.z, grid.x, grid.y,
			      shared_size, threads.x * threads.y * threads.z);
	f<<< grid, threads, shared_size>>>(img->devdata, kern_data_unflipped, out->devdata,
					   img_len, img_wid, kern_len, kern_wid,
					   nkern, nstack, 
					   img_stride_col, img_stride_row, img_stride_stack, img_stride_batch,
					   kern_stride_col_unflipped, kern_stride_row_unflipped,
					   kern_stride_stack, kern_stride_nkern);
        CNDA_THREAD_SYNC;
        hipError_t sts = hipGetLastError();
        if (hipSuccess == sts) 
        {
            work_complete = true;
        }
        else
        {
            if (verbose) printf("threads.x=%i, threads.y=%i, threads.z=%i, grid.x=%i, grid.y=%i,shared_size=%i, nb_threads=%i\n", threads.x, threads.y, threads.z, grid.x, grid.y, shared_size, threads.x * threads.y * threads.z);
            if (verbose) printf("INFO: impl 'conv_patch_stack_reduce' failed (%s), trying next implementation\n",
                                hipGetErrorString(sts));
        }                         
    }

    if (1 && (version==6||version==-1) &&
	!work_complete) //conv_valid_row_reduce
    {
        int outsize = CudaNdarray_SIZE(out);
        int n_blocks = std::min(outsize, NUM_VECTOR_OP_BLOCKS);

	int block_nstack=nstack;
	//Max of 512 threads per blocks.
	//On old hardware, we have a max of 356 threads as we have only 
	//8k registers and the kernel use 23 register
	//TODO: check if we have 8k or 16k of register...
	while(block_nstack*kern_len>320)block_nstack--;
        dim3 n_threads(block_nstack, kern_len, 1);

        int n_reduce_buf = block_nstack * kern_len * sizeof(float);
        /* initial_reduce_boundary is the greatest power of two less than n_reduce_buf/ sizeof(float)
         *
         * if n_reduce_buf == sizeof(float), then initial_reduce_boundary == 0.
         * */
        int initial_reduce_boundary = (1 << (int)(log2((double)(n_reduce_buf/sizeof(float)))));
        if (initial_reduce_boundary == (n_reduce_buf / sizeof(float)))
            initial_reduce_boundary >>= 1;

        if (n_reduce_buf == sizeof(float))
            assert (initial_reduce_boundary == 0);
        else
        {
            assert (initial_reduce_boundary * 2 >= n_reduce_buf/sizeof(float));
            assert (initial_reduce_boundary < n_reduce_buf/sizeof(float));
        }


	void (*f)(int, int, int, int,
		  int, int, int, int, int,
		  float*, int, int, int, int,
		  float*, int, int, int, int,
		  float*, int, int, int, int,
		  int, int, int);

        //std::cerr << "initial_reduce_boundary " << initial_reduce_boundary << "\n";
        //std::cerr << "kerns " << nstack << " " << kern_len << "\n";
        //std::cerr << "n_reduce_buf/sizeof(float) " << n_reduce_buf / sizeof(float) << "\n";
	if(block_nstack==nstack)
	  f=conv_valid_row_reduce<false>;
	else
	  f=conv_valid_row_reduce<true>;
	f<<<n_blocks, n_threads, n_reduce_buf>>>(
                nbatch, nkern, CudaNdarray_HOST_DIMS(img)[1],
                img_len, img_wid,
                kern_len, kern_wid,
                out_len, out_wid,
                img->devdata,
                CudaNdarray_HOST_STRIDES(img)[0], CudaNdarray_HOST_STRIDES(img)[1], 
                img_stride_row, img_stride_col,
                kern->devdata,
                CudaNdarray_HOST_STRIDES(kern)[0], CudaNdarray_HOST_STRIDES(kern)[1],
                CudaNdarray_HOST_STRIDES(kern)[2], CudaNdarray_HOST_STRIDES(kern)[3],
                out->devdata,
                CudaNdarray_HOST_STRIDES(out)[0], CudaNdarray_HOST_STRIDES(out)[1],
                CudaNdarray_HOST_STRIDES(out)[2], CudaNdarray_HOST_STRIDES(out)[3],
                subsample_rows, subsample_cols, initial_reduce_boundary);

        CNDA_THREAD_SYNC;

        hipError_t sts = hipGetLastError();
        if (hipSuccess == sts) 
        {
            work_complete = true;
            if (verbose) printf("INFO: used 'conv_valid_row_reduce' version\n");
        }
        else
        {
            if (verbose) printf("threads.x=%i, threads.y=%i, grid.x=%i, shared_size=%i, nb_threads=%i\n", n_threads.x, n_threads.y, n_blocks, n_reduce_buf, n_threads.x * n_threads.y);
            if (verbose) printf("INFO: impl 'conv_valid_row_reduce' failed (%s), trying next implementation\n",
                    hipGetErrorString(sts));
        }
    }

    if (1 && !work_complete) //conv_reference_valid
    {
        int outsize = CudaNdarray_SIZE(out);
        int n_blocks = std::min(outsize, NUM_VECTOR_OP_BLOCKS);
        int n_threads = std::min(ceil_intdiv(outsize, n_blocks), NUM_VECTOR_OP_THREADS_PER_BLOCK);
        if (0)
        {
            if (verbose) printf("INFO: launching conv_reference_valid\n");
            if (verbose) printf("      img : %i %i %i %i %p  %i %i %i %i\n",
                    nbatch, CudaNdarray_HOST_DIMS(img)[1], img_len, img_wid,
                    img->devdata,
                    CudaNdarray_HOST_STRIDES(img)[0], CudaNdarray_HOST_STRIDES(img)[1], CudaNdarray_HOST_STRIDES(img)[2], CudaNdarray_HOST_STRIDES(img)[3]);
            if (verbose) printf("      kern: %i %i %i %i %p  %i %i %i %i\n", 
                    nkern, nstack, kern_len, kern_wid,
                    kern->devdata,
                    CudaNdarray_HOST_STRIDES(kern)[0], CudaNdarray_HOST_STRIDES(kern)[1], CudaNdarray_HOST_STRIDES(kern)[2], CudaNdarray_HOST_STRIDES(kern)[3]
                        );
            if (verbose) printf("      out : %i %i %i %i %p  %i %i %i %i\n",
                    CudaNdarray_HOST_DIMS(out)[0], CudaNdarray_HOST_DIMS(out)[1], out_len, out_wid,
                    out->devdata,
                    CudaNdarray_HOST_STRIDES(out)[0], CudaNdarray_HOST_STRIDES(out)[1], CudaNdarray_HOST_STRIDES(out)[2], CudaNdarray_HOST_STRIDES(out)[3]);
            if (verbose) printf("   launch params: %i %i %i\n", outsize, n_blocks, n_threads);
        }
        conv_reference_valid<<<n_blocks, n_threads>>>( nbatch, nkern, CudaNdarray_HOST_DIMS(img)[1],
                img_len, img_wid,
                kern_len, kern_wid,
                out_len, out_wid,
                img->devdata, CudaNdarray_HOST_STRIDES(img)[0], CudaNdarray_HOST_STRIDES(img)[1], CudaNdarray_HOST_STRIDES(img)[2], CudaNdarray_HOST_STRIDES(img)[3],
                kern->devdata, CudaNdarray_HOST_STRIDES(kern)[0], CudaNdarray_HOST_STRIDES(kern)[1], CudaNdarray_HOST_STRIDES(kern)[2], CudaNdarray_HOST_STRIDES(kern)[3],
                out->devdata, CudaNdarray_HOST_STRIDES(out)[0], CudaNdarray_HOST_STRIDES(out)[1], CudaNdarray_HOST_STRIDES(out)[2], CudaNdarray_HOST_STRIDES(out)[3],
                subsample_rows, subsample_cols);
        CNDA_THREAD_SYNC;

        hipError_t sts = hipGetLastError();
        if (hipSuccess == sts) 
        {
            work_complete = true;
            if (verbose) printf("INFO: used 'conv_reference_valid' version\n");
        }
        else
        {
            PyErr_Format(PyExc_RuntimeError, "ERROR: all implementations failed! (%s)",
                    hipGetErrorString(sts));
            return -1;
        }
    }
    return 0;

            //PyErr_Format(PyExc_RuntimeError, "Cuda error: %s: %s.\n", "kExp", hipGetErrorString(err));
            //return -1;
}
int 
CudaNdarray_conv_full(const CudaNdarray *img, const CudaNdarray * kern, CudaNdarray * out, int subsample_rows, int subsample_cols, int version = -1, int verbose=0)
{
    const int shared_avail = SHARED_SIZE-150;//144 is the biggest static shared size used with compiling this file.

    int work_complete = 0;
    if (img->nd != 4)
    {
        PyErr_SetString(PyExc_ValueError, "required img of 4D");
        return -1;
    }
    if (kern->nd != 4)
    {
        PyErr_SetString(PyExc_ValueError, "required kern of 4D");
        return -1;
    }
    if (out->nd != 4)
    {
        PyErr_SetString(PyExc_ValueError, "required out of 4D");
        return -1;
    }
    if (0)
    {
        //TODO: rethink these to use physical / logical dimensions, subsampling, offsets, etc.
        assert (CudaNdarray_HOST_DIMS(out)[2] == CudaNdarray_HOST_DIMS(img)[2] + CudaNdarray_HOST_DIMS(kern)[2] - 1);
        assert (CudaNdarray_HOST_DIMS(out)[3] == CudaNdarray_HOST_DIMS(img)[3] + CudaNdarray_HOST_DIMS(kern)[3] - 1);
    }
    assert (CudaNdarray_HOST_DIMS(out)[0] == CudaNdarray_HOST_DIMS(img)[0]);
    assert (CudaNdarray_HOST_DIMS(out)[1] == CudaNdarray_HOST_DIMS(kern)[0]);
    assert (CudaNdarray_HOST_DIMS(img)[1] == CudaNdarray_HOST_DIMS(kern)[1]);

    const int nstack=CudaNdarray_HOST_DIMS(kern)[1];
    const int nbatch=CudaNdarray_HOST_DIMS(img)[0];
    const int nkern=CudaNdarray_HOST_DIMS(kern)[0];
    const int img_wid=CudaNdarray_HOST_DIMS(img)[3];
    const int img_len=CudaNdarray_HOST_DIMS(img)[2];
    const int kern_wid=CudaNdarray_HOST_DIMS(kern)[3];
    const int kern_len=CudaNdarray_HOST_DIMS(kern)[2];
    const int out_wid=CudaNdarray_HOST_DIMS(out)[3];
    const int out_len=CudaNdarray_HOST_DIMS(out)[2];

    const int img_stride_col= CudaNdarray_HOST_STRIDES(img)[3];
    const int img_stride_row=CudaNdarray_HOST_STRIDES(img)[2];
    const int img_stride_stack=CudaNdarray_HOST_STRIDES(img)[1];
    const int img_stride_batch=CudaNdarray_HOST_STRIDES(img)[0];
    const int kern_stride_col= CudaNdarray_HOST_STRIDES(kern)[3];
    const int kern_stride_row=CudaNdarray_HOST_STRIDES(kern)[2];
    const int kern_stride_stack= CudaNdarray_HOST_STRIDES(kern)[1];
    const int kern_stride_nkern=CudaNdarray_HOST_STRIDES(kern)[0];

    const int img_size=img_len*img_wid;
    const int kern_size=kern_len*kern_wid;
    const int out_size=out_len*out_wid;
    const int img_size_byte = img_size*sizeof(float);
    const int kern_size_byte = kern_size*sizeof(float);
    //padded image sizes
    const int img_wid_padded=img_wid+2*kern_wid-2;
    const int img_len_padded=img_len+2*kern_len-2;
    const int img_size_padded=img_len_padded * img_wid_padded;
    const int img_size_padded_byte = img_size_padded*sizeof(float);
    
    //const int out_size_byte = out_size*sizeof(float); // unused 

    bool subsample = subsample_rows!=1 || subsample_cols!=1;

    bool img_contiguous = CudaNdarray_is_c_contiguous(img);
    bool kern_contiguous = CudaNdarray_is_c_contiguous(kern);
    bool out_contiguous = CudaNdarray_is_c_contiguous(out);
    bool c_contiguous = img_contiguous &&  kern_contiguous && out_contiguous;

    bool img_contiguous_2d = (img_stride_col == 1) && (img_stride_row==img_wid);
    bool kern_contiguous_2d = (kern_stride_col == 1) && (kern_stride_row==kern_wid);

    bool img_batch_stack_contiguous = (img_stride_stack==img_stride_row*img_len) && (img_stride_batch==img_stride_stack*nstack);//don't support stride for nbatch and nstack

    //if the lower 2 dims are c_contiguous but flipped, unflipping the stride and not flipping the kernel in shared memroy
    //allow to use a version that use less registers(so is faster)
    //the unflipped version of variable have the original value when we don't need to unflip it, but have the new value when we unflip it.
    bool kern_flipped=true;
    bool kern_contiguous_2d_unflipped = kern_contiguous_2d;
    float * kern_data_unflipped = kern->devdata;
    int kern_stride_col_unflipped=kern_stride_col;
    int kern_stride_row_unflipped=kern_stride_row;
    if(kern_stride_col_unflipped==-1 && kern_stride_row_unflipped==-kern_wid){
      //the last two dimensions are c_contiguous but flipped!
      kern_stride_col_unflipped=1;
      kern_stride_row_unflipped=kern_wid;
      kern_flipped=false;
      kern_contiguous_2d_unflipped = true;
      kern_data_unflipped=&(kern->devdata[(kern_wid-1)*kern_stride_col + (kern_len-1)*kern_stride_row]);
    }

    if (verbose>1)
    {
        printf("INFO: Running conv_full version %d with inputs:\n",version);
        printf("INFO:   img  dim: %i %i %i %i  img  stride: %i %i %i %i\n", 
                CudaNdarray_HOST_DIMS(img)[0], CudaNdarray_HOST_DIMS(img)[1],CudaNdarray_HOST_DIMS(img)[2],CudaNdarray_HOST_DIMS(img)[3],
                CudaNdarray_HOST_STRIDES(img)[0], CudaNdarray_HOST_STRIDES(img)[1],CudaNdarray_HOST_STRIDES(img)[2],CudaNdarray_HOST_STRIDES(img)[3]);
        printf("INFO:   kern dim: %i %i %i %i  kern stride: %i %i %i %i\n",
                CudaNdarray_HOST_DIMS(kern)[0], CudaNdarray_HOST_DIMS(kern)[1],CudaNdarray_HOST_DIMS(kern)[2],CudaNdarray_HOST_DIMS(kern)[3],
                CudaNdarray_HOST_STRIDES(kern)[0], CudaNdarray_HOST_STRIDES(kern)[1],CudaNdarray_HOST_STRIDES(kern)[2],CudaNdarray_HOST_STRIDES(kern)[3]);
    }

    if (!subsample &&
	out_contiguous &&
	(version==3||version==4||version==5||version==-1) &&
	out_wid<512 &&//Maximum of 512 threads by block
	(kern_len+2*kern_len-2)*img_wid_padded*sizeof(float) + kern_size_byte<shared_avail && //their is only 16k of shared memory
	!work_complete) //conv_full_patch_stack_padded
    {
      //version 3 without split
      //version 4 with split (more registers)
      //version 5 with split (more registers) low mem version(some restriction and still more register)
        int nb_split=1;//The number of split (i.e. the number of output pixel each thread compute.)
	if((version==4 || version==5) && out_len>1) nb_split++;//to force the use of split=true when testing.
	if(kern_len==1 && version==5){
	  //version 5 don't support kern_len==1 as 1%0 return -1.
	  version=-1;
	  if(verbose)printf("WARNING:conv full: Asking version 5 with kern_len==1. Combination not supported!\n");
	}
	if(img_size_padded_byte+kern_size_byte>shared_avail) version=5;

	//we pass by ceil_intdiv in case the out_len is not a multiple of nb_split, we want nb_split the number of iteration.
	//Max of 16k of shared memory
	if(version==5)
	  while ((((kern_len+ceil_intdiv(out_len,nb_split)-1)+2*kern_len-2)*img_wid_padded*sizeof(float) + kern_size_byte)>shared_avail) nb_split++;
	
	//327 as we use 25 register
	//version 5 will have only 1 block running at a time, so we can use 32 registers per threads, but their is some other stuff that for the limit to bu lower then 512.
	int max_thread = (version!=5?327:450);
	while (ceil_intdiv(out_len,nb_split)*out_wid>max_thread) nb_split++;
	if(version==-1 && out_size>512)version=4;
	if(version==-1)version=3;


	if(version==-1 && nb_split>1) version=4;
	else if(version==-1) version=3;
	else if(version==3 && nb_split!=1) version=4;//we force version 4 when we need more then 1 split as to be always execute.

	assert(version!=3 || nb_split==1);
	assert(version!=5 || kern_len>1);
	assert(version!=-1);

        dim3 threads(out_wid, ceil_intdiv(out_len,nb_split));
        dim3 grid(nbatch,nkern);

	int shared_size=img_size_padded_byte + kern_size_byte;
	if(version==5)
	  shared_size=((kern_len+threads.y-1)+2*kern_len-2)*img_wid_padded*sizeof(float) + kern_size_byte;
	void (*f)(float*, float*, float*,
		  int, int, int, int,
		  int, int, int, int,
		  int, int, int, int,
		  int, int);

#define CONV_FULL_PATCH_STACK_PADDED_SPECIAL(kern_wid) \
             if(img_contiguous_2d && kern_contiguous_2d_unflipped && version==3 && kern_flipped) f=conv_full_patch_stack_padded<true,kern_wid,true,false,false>;\
	else if(img_contiguous_2d && kern_contiguous_2d_unflipped && version==4 && kern_flipped) f=conv_full_patch_stack_padded<true,kern_wid,true,true,false>;\
	else if(img_contiguous_2d && kern_contiguous_2d_unflipped && version==5 && kern_flipped) f=conv_full_patch_stack_padded<true,kern_wid,true,false,true>;\
	else if(version==3 && kern_flipped) f=conv_full_patch_stack_padded<true,kern_wid,false,false,false>;\
	else if(version==4 && kern_flipped)f=conv_full_patch_stack_padded<true,kern_wid,false,true,false>;\
	else if(version==5 && kern_flipped)f=conv_full_patch_stack_padded<true,kern_wid,false,false,true>;\
	else if(img_contiguous_2d && kern_contiguous_2d_unflipped && version==3) f=conv_full_patch_stack_padded<false,kern_wid,true,false,false>;\
	else if(img_contiguous_2d && kern_contiguous_2d_unflipped && version==4) f=conv_full_patch_stack_padded<false,kern_wid,true,true,false>;\
	else if(img_contiguous_2d && kern_contiguous_2d_unflipped && version==5) f=conv_full_patch_stack_padded<false,kern_wid,true,false,true>;\
	else if(version==3) f=conv_full_patch_stack_padded<false,kern_wid,false,false,false>;\
	else if(version==4) f=conv_full_patch_stack_padded<false,kern_wid,false,true,false>;\
	else if(version==5) f=conv_full_patch_stack_padded<false,kern_wid,false,false,true>;\
	else assert(false);

	switch(kern_wid){
#ifdef UNROLL_LOOP
         case 1: CONV_FULL_PATCH_STACK_PADDED_SPECIAL(1); break;
         case 2: CONV_FULL_PATCH_STACK_PADDED_SPECIAL(2); break;
         case 3: CONV_FULL_PATCH_STACK_PADDED_SPECIAL(3); break;
         case 4: CONV_FULL_PATCH_STACK_PADDED_SPECIAL(4); break;
         case 5: CONV_FULL_PATCH_STACK_PADDED_SPECIAL(5); break;//test_conv.py:test_full
         case 6: CONV_FULL_PATCH_STACK_PADDED_SPECIAL(6); break;//test_conv.py:test_full
         case 7: CONV_FULL_PATCH_STACK_PADDED_SPECIAL(7); break;//test_nnet.py:test_lenet_64
         case 8: CONV_FULL_PATCH_STACK_PADDED_SPECIAL(8); break;//test_conv.py:test_full
         case 9: CONV_FULL_PATCH_STACK_PADDED_SPECIAL(9); break;//test_nnet.py:test_lenet_256
         case 10: CONV_FULL_PATCH_STACK_PADDED_SPECIAL(10); break;//test_conv.py:test_full
         case 12: CONV_FULL_PATCH_STACK_PADDED_SPECIAL(12); break;//test_conv.py:test_full
                  //////// Special cases
         case 28: CONV_FULL_PATCH_STACK_PADDED_SPECIAL(28); break;
#endif
                  //////// default case
	 default:
	   if(!msgdisplayed_conv_full_patch_stack__kern_width){
	     printf("OPTIMISATION HINT: conv_full_patch_stack_padded template default add kern_wid=%d in %s at line %i to have an optimized version for your kern_wid\n", kern_wid, __FILE__, __LINE__);
	     msgdisplayed_conv_full_patch_stack__kern_width = true;
	   }
           CONV_FULL_PATCH_STACK_PADDED_SPECIAL(0);
	}


	f<<< grid, threads, shared_size>>>
	     (img->devdata, kern_data_unflipped, out->devdata,
	      img_len, img_wid, kern_len, kern_wid, nkern, nstack,
	      img_stride_col, img_stride_row, img_stride_stack,
	      img_stride_batch, kern_stride_col_unflipped, kern_stride_row_unflipped,
	      kern_stride_stack, kern_stride_nkern);

        CNDA_THREAD_SYNC;
        hipError_t sts = hipGetLastError();
        if (hipSuccess == sts) 
        {
            if (verbose>1) printf("threads.x=%i, threads.y=%i, grid.x=%i, grid.y=%i,shared_size=%i, nb_threads=%i, out_len=%i, nb_split=%i, version=%i\n", threads.x, threads.y, grid.x, grid.y, shared_size, threads.x * threads.y, out_len, nb_split, version);
            if (verbose) printf("INFO: used 'conv_full_patch_stack_padded' nb_split=%d low_mem=%s\n",nb_split,(version==5?"true":"false"));
            work_complete = true;
        }
        else
        {
            if (verbose) printf("threads.x=%i, threads.y=%i, grid.x=%i, grid.y=%i,shared_size=%i, nb_threads=%i, out_len=%i, nb_split=%i, version=%i\n", threads.x, threads.y, grid.x, grid.y, shared_size, threads.x * threads.y, out_len, nb_split, version);
            if (verbose) printf("INFO: impl 'conv_full_patch_stack_padded' %s %s failed (%s), trying next implementation\n",
				version==3?"no split": "split",(version==5?"low_mem":"not_low_mem"),
                                hipGetErrorString(sts));
        }                         
    }

    if (!subsample && c_contiguous &&
	(version==0||version==-1) &&
	out_size<512 &&//Maximum of 512 theads by block
	nstack == 1 &&// don't implement the stack in the kernel.
	img_size_byte+kern_size_byte<shared_avail && //their is only 16k of shared memory
	!work_complete) //conv_full_patch
    {
        dim3 threads(out_wid, out_len);
        dim3 grid(nbatch,nkern);
        int shared_size=(img_size + kern_size)*sizeof(float);
	//TODO assert c_continious for img, kern and out in the 2 inner dimensions.

	conv_full_patch<<< grid, threads, shared_size>>>
	  (img->devdata,
	   kern->devdata,
	   out->devdata,
	   img_len, img_wid,
	   kern_len, kern_wid,
	   nkern, nstack);

        CNDA_THREAD_SYNC;
        hipError_t sts = hipGetLastError();
        if (hipSuccess == sts) 
        {
            if (verbose) printf("INFO: used 'conv_full_patch' version\n");
            work_complete = true;
        }
        else
        {
            if (verbose) printf("threads.x=%i, threads.y=%i, grid.x=%i, grid.y=%i, shared_size=%i, nb_threads=%i\n", threads.x, threads.y, grid.x, grid.y, shared_size, threads.x * threads.y);
            if (verbose) printf("INFO: impl 'conv_full_patch' failed (%s), trying next implementation\n",
                                hipGetErrorString(sts));
        }                         
    }
    if (false && !subsample && //disabled as test fail for this kernel
	(version==1||version==-1) &&
	out_size<512 &&//Maximum of 512 theads by block
        (nbatch > 20 || version==1) &&  // we only launch nbatch blocks, so make sure there is enough to be worth it, but if we specify the version, this check should not be done to allow testing.
	nstack*img_size_byte+nstack*kern_size_byte<shared_avail && //there is only 16k of shared memory
	!work_complete) //conv_full_load_everything
    {
        dim3 threads(out_wid, out_len);
        dim3 grid(nbatch);
        int shared_size=(img_size + kern_size)*nstack*sizeof(float);
	//TODO assert c_continious for img, kern and out in the 2 inner dimensions.

        //typeof(conv_full_load_everything<0>) f = ;
	void (*f)(float*, float*, float*,
		  int, int, int, int, int, int,
		  int, int, int, int, int, int, int, int) = conv_full_load_everything<0>;

        switch(nstack)
        {
#ifdef UNROLL_LOOP
            case 1: f = conv_full_load_everything<1>; break;
            //case 10: f = conv_full_load_everything<10>; break;
            //case 30: f = conv_full_load_everything<30>; break;  //This is actually slower than the general version??
#endif
	    default:
	      printf("OPTIMISATION HINT: conv_full_load_everything template default add kern_wid=%d in %s at line %i to have an optimized version for your kern_wid\n", kern_wid, __FILE__, __LINE__);
	      f = conv_full_load_everything<0>;
        };

	f<<< grid, threads, shared_size>>>
	  (img->devdata,
	   kern->devdata,
	   out->devdata,
	   img_len, img_wid, 
	   kern_len, kern_wid,
	   nkern, nstack,
           CudaNdarray_HOST_STRIDES(img)[3],
           CudaNdarray_HOST_STRIDES(img)[2],
           CudaNdarray_HOST_STRIDES(img)[1],
           CudaNdarray_HOST_STRIDES(img)[0],
           CudaNdarray_HOST_STRIDES(kern)[3],
           CudaNdarray_HOST_STRIDES(kern)[2],
           CudaNdarray_HOST_STRIDES(kern)[1],
           CudaNdarray_HOST_STRIDES(kern)[0]
           );

        CNDA_THREAD_SYNC;
        hipError_t sts = hipGetLastError();
        if (hipSuccess == sts) 
        {
            if (verbose) printf("INFO: used 'conv_full_load_everything' version\n");
            work_complete = true;
        }
        else
        {
            if (verbose) printf("threads.x=%i, threads.y=%i, grid.x=%i, grid.y=%i, shared_size=%i, nb_threads=%i\n", threads.x, threads.y, grid.x, grid.y, shared_size, threads.x * threads.y);
            if (verbose) printf("INFO: impl 'conv_full_load_everything' failed (%s), trying next implementation\n",
                                hipGetErrorString(sts));
        }                         
    }

    if (!subsample &&
	img_batch_stack_contiguous &&
	out_contiguous &&
	(version==2||version==-1) &&
	out_size<512 &&//Maximum of 512 theads by block
	img_size_byte+kern_size_byte<shared_avail && //their is only 16k of shared memory
	!work_complete) //conv_full_patch_stack
    {
        dim3 threads(out_wid, out_len);
        dim3 grid(nbatch,nkern);
        int shared_size=(img_size + kern_size)*sizeof(float);

	void (*f)(float*, float*, float*,
		  int, int, int, int,
		  int, int, int, int,
		  int, int, int, int);

        if(img_contiguous_2d && kern_contiguous_2d) f=conv_full_patch_stack<true,true>;\
        else if(img_contiguous_2d && !kern_contiguous_2d) f=conv_full_patch_stack<true,false>;\
        else if(!img_contiguous_2d && kern_contiguous_2d) f=conv_full_patch_stack<false,true>;\
        else if(!img_contiguous_2d && !kern_contiguous_2d) f=conv_full_patch_stack<false,false>;

        f<<< grid, threads, shared_size>>>(
                img->devdata,
                kern->devdata,
                out->devdata,
                img_len, img_wid,
                kern_len, kern_wid,
                nkern, nstack,img_stride_col, img_stride_row,
                kern_stride_col, kern_stride_row,
                kern_stride_stack, kern_stride_nkern);
        CNDA_THREAD_SYNC;
        hipError_t sts = hipGetLastError();
        if (hipSuccess == sts) 
        {
            if (verbose) printf("INFO: used 'conv_full_patch_stack' version\n");
            work_complete = true;
        }
        else
        {
            if (verbose) printf("threads.x=%i, threads.y=%i, grid.x=%i, grid.y=%i, shared_size=%i, nb_threads=%i\n", threads.x, threads.y, grid.x, grid.y, shared_size, threads.x * threads.y);
            if (verbose) printf("INFO: impl 'conv_full_patch_stack' failed (%s), trying next implementation\n",
                                hipGetErrorString(sts));
        }                         
    }
    if (1 && !work_complete) //conv_reference_full
    {
        if(verbose>1)printf("INFO: will start conv_reference_full\n");

        int outsize = CudaNdarray_SIZE(out);
        int n_blocks = std::min(outsize, NUM_VECTOR_OP_BLOCKS);
        int n_threads = std::min(ceil_intdiv(outsize, n_blocks), NUM_VECTOR_OP_THREADS_PER_BLOCK);
        if (0)
        {
            if (verbose) printf("INFO: launching conv_reference_valid\n");
            if (verbose) printf("      img : %i %i %i %i %p  %i %i %i %i\n",
                    CudaNdarray_HOST_DIMS(img)[0], CudaNdarray_HOST_DIMS(img)[1], CudaNdarray_HOST_DIMS(img)[2], CudaNdarray_HOST_DIMS(img)[3],
                    img->devdata,
                    CudaNdarray_HOST_STRIDES(img)[0], CudaNdarray_HOST_STRIDES(img)[1], CudaNdarray_HOST_STRIDES(img)[2], CudaNdarray_HOST_STRIDES(img)[3]);
            if (verbose) printf("      kern: %i %i %i %i %p  %i %i %i %i\n", 
                    CudaNdarray_HOST_DIMS(kern)[0], CudaNdarray_HOST_DIMS(kern)[1], CudaNdarray_HOST_DIMS(kern)[2], CudaNdarray_HOST_DIMS(kern)[3],
                    kern->devdata,
                    CudaNdarray_HOST_STRIDES(kern)[0], CudaNdarray_HOST_STRIDES(kern)[1], CudaNdarray_HOST_STRIDES(kern)[2], CudaNdarray_HOST_STRIDES(kern)[3]
                        );
            if (verbose) printf("      out : %i %i %i %i %p  %i %i %i %i\n",
                    CudaNdarray_HOST_DIMS(out)[0], CudaNdarray_HOST_DIMS(out)[1], CudaNdarray_HOST_DIMS(out)[2], CudaNdarray_HOST_DIMS(out)[3],
                    out->devdata,
                    CudaNdarray_HOST_STRIDES(out)[0], CudaNdarray_HOST_STRIDES(out)[1], CudaNdarray_HOST_STRIDES(out)[2], CudaNdarray_HOST_STRIDES(out)[3]);
            if (verbose) printf("   launch params: %i %i %i\n", outsize, n_blocks, n_threads);
            if (verbose) printf("   subsample params: %i %i\n", subsample_rows, subsample_cols);
        }
        conv_reference_full<<<n_blocks, n_threads>>>(CudaNdarray_HOST_DIMS(img)[0], CudaNdarray_HOST_DIMS(kern)[0], CudaNdarray_HOST_DIMS(img)[1],
                CudaNdarray_HOST_DIMS(img)[2], CudaNdarray_HOST_DIMS(img)[3],
                CudaNdarray_HOST_DIMS(kern)[2], CudaNdarray_HOST_DIMS(kern)[3],
                CudaNdarray_HOST_DIMS(out)[2], CudaNdarray_HOST_DIMS(out)[3],
                img->devdata, CudaNdarray_HOST_STRIDES(img)[0], CudaNdarray_HOST_STRIDES(img)[1], CudaNdarray_HOST_STRIDES(img)[2], CudaNdarray_HOST_STRIDES(img)[3],
                kern->devdata, CudaNdarray_HOST_STRIDES(kern)[0], CudaNdarray_HOST_STRIDES(kern)[1], CudaNdarray_HOST_STRIDES(kern)[2], CudaNdarray_HOST_STRIDES(kern)[3],
                out->devdata, CudaNdarray_HOST_STRIDES(out)[0], CudaNdarray_HOST_STRIDES(out)[1], CudaNdarray_HOST_STRIDES(out)[2], CudaNdarray_HOST_STRIDES(out)[3],
                subsample_rows, subsample_cols);
        CNDA_THREAD_SYNC;

        hipError_t sts = hipGetLastError();
        if (hipSuccess == sts) 
        {
            if (verbose) printf("INFO: used 'conv_reference_full' version ishp(%d, %d) kshp(%d, %d) oshp(%d, %d) nbatch=%d nkern=%d nstack=%d subsample=%d\n",
				img_len,img_wid, kern_len, kern_wid,
				out_len, out_wid, nbatch, nkern, nstack, subsample);
            work_complete = true;
        }
        else
        {
	  if (verbose) printf("threads.x=%i, threads.y=%i, grid.x=%i, grid.y=%i, shared_size=%i, nb_threads=%i\n", n_threads, 1, n_blocks, 1, 0, n_threads);
	  if (verbose) printf("INFO: impl 'conv_reference_full' failed (%s), trying next implementation\n",
			      hipGetErrorString(sts));
	  PyErr_Format(PyExc_RuntimeError, "ERROR: all implementations failed! (%s)",
		       hipGetErrorString(sts));
            return -1;
        }
    }
    return 0;
}

PyObject * 
CudaNdarray_Conv(const CudaNdarray *img, const CudaNdarray * kern,
		 CudaNdarray * out, const int mode,
		 const int subsample_rows, const int subsample_cols,
		 const int version, const int verbose)
{
    if (img->nd != 4) { PyErr_SetString(PyExc_ValueError, "CudaNdarray 4-D tensor required"); return NULL;}
    if (kern->nd != 4) { PyErr_SetString(PyExc_ValueError, "CudaNdarray 4-D tensor required"); return NULL;}

    int out_dim[4];
    out_dim[0] = CudaNdarray_HOST_DIMS(img)[0];
    out_dim[1] = CudaNdarray_HOST_DIMS(kern)[0];
    int logical_rows, logical_cols;
    if (mode == ConvMode_VALID)
    {
        logical_rows = CudaNdarray_HOST_DIMS(img)[2] - CudaNdarray_HOST_DIMS(kern)[2] + 1;
        logical_cols = CudaNdarray_HOST_DIMS(img)[3] - CudaNdarray_HOST_DIMS(kern)[3] + 1;
    }
    else
    {
        logical_rows = CudaNdarray_HOST_DIMS(img)[2] + CudaNdarray_HOST_DIMS(kern)[2] - 1;
        logical_cols = CudaNdarray_HOST_DIMS(img)[3] + CudaNdarray_HOST_DIMS(kern)[3] - 1;
    }
    out_dim[2] = ceil_intdiv(logical_rows, subsample_rows);
    out_dim[3] = ceil_intdiv(logical_cols, subsample_cols);
    
    CudaNdarray * rval = out;
    if(!(out && out->nd==4 && CudaNdarray_is_c_contiguous(out) 
	 && CudaNdarray_HOST_DIMS(out)[0]==out_dim[0]
	 && CudaNdarray_HOST_DIMS(out)[1]==out_dim[1]
	 && CudaNdarray_HOST_DIMS(out)[2]==out_dim[2]
	 && CudaNdarray_HOST_DIMS(out)[3]==out_dim[3])){
      if (out)
      {
          fprintf(stderr, "Warning: Conv is ignoring 'out' argument with wrong structure.\n");
      }
      rval = (CudaNdarray*)CudaNdarray_NewDims(4,out_dim);
    }
    if ((rval==NULL) 
            || ((mode==ConvMode_VALID) && CudaNdarray_conv_valid(img, kern, rval, subsample_rows, subsample_cols, version, verbose))
            || ((mode==ConvMode_FULL) && CudaNdarray_conv_full(img, kern, rval, subsample_rows, subsample_cols, version, verbose))
            )
    {
        // if rval is something we just allocated,
        // and there was a problem, then we have to free it.
        if (rval != out) Py_XDECREF(rval);
        return NULL;
    }
    //TODO: Get refcount story clearer!
    //      This function does a weird thing as work-around with Conv_VARARGS
    if (rval == out) Py_INCREF(rval);
    return (PyObject*)rval;
}

